#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>
#include <limits>
#include <stdexcept>
#include <fstream>
#include <math.h>
#include <iomanip>
#include <cstdlib>
#include <random>
#include <algorithm>
#include <cstring>

#include <filesystem>

//#include "fastPRNG.h"

//#include <Random123/philox.h>             // first, so mulhilo32 is chosen only once
//#include <Random123/uniform.hpp>

//#include "random123/philox_uniform.cuh"
//using fastPRNG;
//fastXS64 fastR;

#include "parallel_pso.cuh"
#include "dual_numbers.cuh"
/*
namespace dual {

class DualNumber {
public:
    double real;
    double dual;

    __host__ __device__ DualNumber(double real = 0.0, double dual = 0.0) : real(real), dual(dual) {}

    __host__ __device__ DualNumber& operator+=(const DualNumber& rhs) {
        real += rhs.real;
        dual += rhs.dual;
        return *this;
    }

    __host__ __device__ DualNumber operator+(const DualNumber& rhs) const {
        return DualNumber(real + rhs.real, dual + rhs.dual);
    }

    __host__ __device__ DualNumber operator-(const DualNumber& rhs) const {
        return DualNumber(real - rhs.real, dual - rhs.dual);
    }

    __host__ __device__ DualNumber operator*(const DualNumber& rhs) const {
        return DualNumber(real * rhs.real, dual * rhs.real + real * rhs.dual);
    }

    __host__ __device__ DualNumber operator/(const DualNumber& rhs) const {
        double denom = rhs.real * rhs.real;
        return DualNumber(real / rhs.real, (dual * rhs.real - real * rhs.dual) / denom);
    }
    // operator for double - DualNumber
    __host__ __device__ friend DualNumber operator-(double lhs, const DualNumber& rhs) {
        return DualNumber(lhs - rhs.real, -rhs.dual);
    }

    // operator for double * DualNumber
    __host__ __device__ friend DualNumber operator*(double lhs, const DualNumber& rhs) {
        return DualNumber(lhs * rhs.real, lhs * rhs.dual);
    }
};

__host__ __device__ inline dual::DualNumber dual_abs(const dual::DualNumber &a) {
    return (a.real < 0.0) ? dual::DualNumber(-a.real, -a.dual) : a;
}

__host__ __device__ DualNumber sin(const DualNumber& x) {
    return DualNumber(sinf(x.real), x.dual * cosf(x.real));
}

__host__ __device__ DualNumber cos(const DualNumber& x) {
    return DualNumber(cosf(x.real), -x.dual * sinf(x.real));
}

__host__ __device__ DualNumber exp(const DualNumber& x) {
    double ex = expf(x.real);
    return DualNumber(ex, x.dual * ex);
}

__host__ __device__ DualNumber sqrt(const DualNumber& x) {
    double sr = sqrtf(x.real);
    return DualNumber(sr, x.dual / (2.0 * sr));
}

__host__ __device__ DualNumber atan2(const DualNumber& y, const DualNumber& x) {
    double denom = x.real * x.real + y.real * y.real;
    return DualNumber(atan2f(y.real, x.real), (x.real * y.dual - y.real * x.dual) / denom);
}

template<typename T>
__host__ __device__ T pow(const T& base, double exponent) {
    return T(powf(base.real, exponent), exponent * powf(base.real, exponent - 1) * base.dual);
}

} // end of dual
*/


namespace util {

__device__ double dot_product_device(const double* a, const double* b, int size) {
    double sum = 0.0;
    for (int i = 0; i < size; ++i) {
        sum += a[i] * b[i];
    }
    return sum;
}

__device__ void outer_product_device(const double* v1, const double* v2, double* result, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
	    int idx = i * size + j;
	    if (idx < size*size){
		result[idx] = v1[i] * v2[j];    
            } else {
   	        printf("outer product out of bounds..\ndim=%d i*size+j=%d\n",size, i * size + j);
            }
	}
    }
}

template<int DIM>
__device__ double calculate_gradient_norm(const double* g) {
    double grad_norm = 0.0;
    for (int i = 0; i < DIM; ++i) {
        grad_norm += g[i] * g[i];
    }
    return sqrt(grad_norm);
}

template<int DIM>
__device__ void compute_search_direction(double* p,const double*  H,const double* g) {
    for (int i = 0; i < DIM; i++) {
        double sum=0.0;
        for (int j = 0; j < DIM; j++) {
           sum += H[i * DIM + j] * g[j]; // i * dim + j since H is flattened arr[]
        }    
    p[i] = -sum;
    }
}

// wrap kernel definitions extern "C" block so that their symbols are exported with C linkage
extern "C" {
__device__ __noinline__ void vector_add(const double* a, const double* b, double* result, int size) {
    for (int i = 0; i < size; ++i) {
        result[i] = a[i] + b[i];
    }
}

__device__ __noinline__ void vector_scale(const double* a, double scalar, double* result, int dim) {
    for (int i = 0; i < dim; ++i) {
        result[i] = a[i] * scalar;
    }
}
}// end extern C

__device__ void initialize_identity_matrix(double* H, int dim) {
    for (int i = 0; i < dim; ++i) {
        for (int j = 0; j < dim; ++j) {
            H[i * dim + j] = (i == j) ? 1.0 : 0.0;
        }
    }
}

__device__ bool valid(double x) {
    if (isinf(x)) {
        return false;
    } else if (isnan(x)) {
	return false;
    } else {
        return true;
    }
}


__device__ double pow2(double x) {
    return x * x;
}


__device__ void initialize_identity_matrix_device(double* H, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            H[i * n + j] = (i == j) ? 1.0 : 0.0;
        }
    }
}

template<int DIM>
__device__ void matrix_multiply_device(const double* A, const double* B, double* C) {
    for (int i = 0; i < DIM; ++i) {
        for (int j = 0; j < DIM; ++j) {
            double sum = 0.0;
            for (int k = 0; k < DIM; ++k) {
                sum += A[i * DIM + k] * B[k * DIM + j];
            }
            C[i * DIM + j] = sum;
        }
    }
}


// BFGS update with compile-time dimension
template<int DIM>
__device__ void bfgs_update(double* H, const double* s, const double* y, double sTy) {
    if (::fabs(sTy) < 1e-14) return;
    double rho = 1.0 / sTy;
    
    // Compute H_new element-wise without allocating large temporary matrices.
    // H_new = (I - rho * s * y^T) * H * (I - rho * y * s^T) + rho * s * s^T
    double H_new[DIM * DIM];  // Temporary array (DIM^2 elements)
    
    for (int i = 0; i < DIM; i++) {
        for (int j = 0; j < DIM; j++) {
            double sum = 0.0;
            for (int k = 0; k < DIM; k++) {
                // Compute element (i,k) of (I - rho * s * y^T)
                double A_ik = ((i == k) ? 1.0 : 0.0) - rho * s[i] * y[k];
                double inner = 0.0;
                for (int m = 0; m < DIM; m++) {
                    // Compute element (m,j) of (I - rho * y * s^T)
                    double B_mj = ((m == j) ? 1.0 : 0.0) - rho * y[m] * s[j];
                    inner += H[k * DIM + m] * B_mj;
                }
                sum += A_ik * inner;
            }
            // Add the rho * s * s^T term
            H_new[i * DIM + j] = sum + rho * s[i] * s[j];
        }
    }
    
    // Copy H_new back into H
    for (int i = 0; i < DIM * DIM; i++) {
        H[i] = H_new[i];
    }
}


// function to calculate scalar directional direvative d = g * p
__device__ double directional_derivative(const double *grad, const double *p, int dim) {
    double d = 0.0;
    for (int i = 0; i < dim; ++i) {
        d += grad[i] * p[i];
    }
    return d;
}

/*
template<int DIM>
__device__
dual::DualNumber rosenbrock(const dual::DualNumber* x) {
    dual::DualNumber sum = 0.0;
    for (int i = 0; i < DIM - 1; ++i) {
	sum += dual::pow(1 - x[i],2) + 100 * dual::pow(x[i+1] - dual::pow(x[i], 2), 2); 
        //sum = sum + 100 * (x[i + 1] - x[i] * x[i]) * (x[i + 1] - x[i] * x[i]) + (1 - x[i]) * (1 - x[i]);
    }
    return sum;
}

template<int DIM>
__host__ __device__
double rosenbrock(const double* x) {
    double sum = 0.0;
    for (int i = 0; i < DIM - 1; ++i) {
        sum += 100 * (x[i + 1] - x[i] * x[i]) * (x[i + 1] - x[i] * x[i]) + (1 - x[i]) * (1 - x[i]);
    }
    return sum;
}

template<int DIM>
__device__
dual::DualNumber rastrigin(const dual::DualNumber* x) {
    const double A = 10.0;
    dual::DualNumber sum(A * DIM, 0.0);
    for(int i=0; i<DIM; i++){
	dual::DualNumber xi_sq = x[i] * x[i];
	dual::DualNumber cterm = cos(2.0 * M_PI * x[i]);
        sum = sum + (xi_sq - A * cterm);
    }
    return sum;
}

template<int DIM>
__host__ __device__
double rastrigin(const double* x) {
    const double A = 10.0;
    double val = A * DIM;
    for(int i=0; i<DIM; i++){
        double xi = x[i];
        val += xi*xi - A*::cosf(2.0f*M_PI*xi);
    }
    return val;
    
    	double sum = 10 * DIM;
    for (int i = 0; i < DIM; ++i) {
        sum += x[i] * x[i] - 10 * cos(2 * M_PI * x[i]);
    }
    return sum;
*/


/*
template<int DIM>
struct Rosenbrock {
    __device__ static dual::DualNumber evaluate(const dual::DualNumber* x) {
        //printf("in dualrosen");
        return rosenbrock<DIM>(x);
    }

    __host__ __device__ static double evaluate(const double* x) {
        return rosenbrock<DIM>(x);
    }
};

template<int DIM>
struct Rastrigin {
    __device__ static dual::DualNumber evaluate(const dual::DualNumber* x) {
        return rastrigin<DIM>(x);
    }

    __host__ __device__ static double evaluate(const double* x) {
        return rastrigin<DIM>(x);
    }
};
*/

// Ackley Function (general d-dimensions)
//   f(x) = -20 exp\Bigl(-0.2\sqrt{\frac{1}{d}\sum_{i=1}^{d}x_i^2}\Bigr)
//          - exp\Bigl(\frac{1}{d}\sum_{i=1}^{d}\cos(2\pi x_i)\Bigr)
//          + 20 + e
/*template<int DIM>
__device__
dual::DualNumber ackley(const dual::DualNumber* x) {
    dual::DualNumber sum_sq = 0.0;
    dual::DualNumber sum_cos = 0.0;
    for (int i = 0; i < DIM; ++i) {
        sum_sq += dual::pow(x[i], 2);
        sum_cos += dual::cos(2.0 * M_PI * x[i]);
    }
    dual::DualNumber term1 = dual::DualNumber(-20.0) * dual::exp(-0.2 * dual::sqrt(sum_sq / DIM));
    dual::DualNumber term2 = dual::DualNumber(0.0) - dual::exp(sum_cos / DIM);
    return term1 + term2 + 20.0 + dual::exp(1.0);
}

template<int DIM>
__host__ __device__
double ackley(const double* x) {
    double sum_sq = 0.0;
    double sum_cos = 0.0;
    for (int i = 0; i < DIM; ++i) {
        sum_sq += x[i] * x[i];
        sum_cos += cos(2.0 * M_PI * x[i]);
    }
    double term1 = -20.0 * exp(-0.2 * sqrt(sum_sq / DIM));
    double term2 = -exp(sum_cos / DIM);
    return term1 + term2 + 20.0 + exp(1.0);
}

template<int DIM>
struct Ackley {
    __device__ static dual::DualNumber evaluate(const dual::DualNumber* x) {
        return ackley<DIM>(x);
    }
    __host__ __device__ static double evaluate(const double* x) {
        return ackley<DIM>(x);
    }
};*/

// Goldstein-Price Function
//   f(x,y) = [1+(x+y+1)^2 (19-14x+3x^2-14y+6xy+3y^2)]
//            [30+(2x-3y)^2 (18-32x+12x^2+48y-36xy+27y^2)]
template<int DIM>
__device__
dual::DualNumber goldstein_price(const dual::DualNumber* x) {
    static_assert(DIM == 2, "Goldstein-Price is defined for 2 dimensions only.");
    dual::DualNumber x1 = x[0];
    dual::DualNumber x2 = x[1];
    dual::DualNumber term1 = dual::DualNumber(1.0) + dual::pow(x1 + x2 + 1.0, 2) *
        (19.0 - 14.0 * x1 + 3.0 * dual::pow(x1, 2) - 14.0 * x2 + 6.0 * x1 * x2 + 3.0 * dual::pow(x2, 2));
    dual::DualNumber term2 = dual::DualNumber(30.0) + dual::pow(2.0 * x1 - 3.0 * x2, 2) *
        (18.0 - 32.0 * x1 + 12.0 * dual::pow(x1, 2) + 48.0 * x2 - 36.0 * x1 * x2 + 27.0 * dual::pow(x2, 2));
    return term1 * term2;
}

template<int DIM>
__host__ __device__
double goldstein_price(const double* x) {
    static_assert(DIM == 2, "Goldstein-Price is defined for 2 dimensions only.");
    double x1 = x[0];
    double x2 = x[1];
    double term1 = 1.0 + pow(x1 + x2 + 1.0, 2) *
        (19.0 - 14.0 * x1 + 3.0 * pow(x1, 2) - 14.0 * x2 + 6.0 * x1 * x2 + 3.0 * pow(x2, 2));
    double term2 = 30.0 + pow(2.0 * x1 - 3.0 * x2, 2) *
        (18.0 - 32.0 * x1 + 12.0 * pow(x1, 2) + 48.0 * x2 - 36.0 * x1 * x2 + 27.0 * pow(x2, 2));
    return term1 * term2;
}

template<int DIM>
struct GoldsteinPrice {
    __device__ static dual::DualNumber evaluate(const dual::DualNumber* x) {
        return goldstein_price<DIM>(x);
    }
    __host__ __device__ static double evaluate(const double* x) {
        return goldstein_price<DIM>(x);
    }
};

// Eggholder Function
//   f(x,y) = -(y+47) sin\Bigl(\sqrt{\Bigl|x/2+y+47\Bigr|}\Bigr)
//            - x sin\Bigl(\sqrt{\Bigl|x-(y+47)\Bigr|}\Bigr)

template<int DIM>
__device__ dual::DualNumber eggholder(const dual::DualNumber* x) {
    static_assert(DIM == 2, "Eggholder is defined for 2 dimensions only.");
    dual::DualNumber x1 = x[0], x2 = x[1];
    // Use (0 - value) in place of unary minus
    dual::DualNumber term1 = (dual::DualNumber(0.0) - (x2 + dual::DualNumber(47.0))) *
        dual::sin(dual::sqrt(dual_abs(x1 / dual::DualNumber(2.0) + x2 + dual::DualNumber(47.0))));
    dual::DualNumber term2 = (dual::DualNumber(0.0) - x1) *
        dual::sin(dual::sqrt(dual_abs(x1 - (x2 + dual::DualNumber(47.0)))));
    return term1 + term2;
}

template<int DIM>
__device__ double eggholder(const double* x) {
    static_assert(DIM == 2, "Eggholder is defined for 2 dimensions only.");
    double x1 = x[0];
    double x2 = x[1];
    double term1 = -(x2 + 47.0) *
        sin(sqrt(fabs(x1 / 2.0 + x2 + 47.0)));
    double term2 = -x1 *
        sin(sqrt(fabs(x1 - (x2 + 47.0))));
    return term1 + term2;
}

template<int DIM>
struct Eggholder {
    __host__ __device__ static dual::DualNumber evaluate(const dual::DualNumber* x) {
        return eggholder<DIM>(x);
    }
    __host__ __device__ static double evaluate(const double* x) {
        return eggholder<DIM>(x);
    }
};


// Himmelblau's Function (2D only)
template<int DIM>
__device__
dual::DualNumber himmelblau(const dual::DualNumber* x) {
    static_assert(DIM == 2, "Himmelblau's function is defined for 2 dimensions only.");
    dual::DualNumber x1 = x[0], x2 = x[1];
    dual::DualNumber term1 = dual::pow(x1 * x1 + x2 - dual::DualNumber(11.0), 2);
    dual::DualNumber term2 = dual::pow(x1 + x2 * x2 - dual::DualNumber(7.0), 2);
    return term1 + term2;
}

template<int DIM>
__host__ __device__
double himmelblau(const double* x) {
    static_assert(DIM == 2, "Himmelblau's function is defined for 2 dimensions only.");
    double x1 = x[0], x2 = x[1];
    double term1 = pow(x1 * x1 + x2 - 11.0, 2);
    double term2 = pow(x1 + x2 * x2 - 7.0, 2);
    return term1 + term2;
}

template<int DIM>
struct Himmelblau {
    __host__ __device__ static dual::DualNumber evaluate(const dual::DualNumber* x) {
        return himmelblau<DIM>(x);
    }
    __host__ __device__ static double evaluate(const double* x) {
        return himmelblau<DIM>(x);
    }
};

/*template<typename Function, int DIM>
__device__ void calculateGradientUsingAD(double *x, double *gradient) {
    dual::DualNumber xDual[DIM];

    for (int i = 0; i < DIM; ++i) { // // iterate through each dimension (vairbale)
        xDual[i] = dual::DualNumber(x[i], 0.0);
    }

    // calculate the partial derivative of  each dimension
    for (int i = 0; i < DIM; ++i) {
        xDual[i].dual = 1.0; // derivative w.r.t. dimension i
        dual::DualNumber result = Function::evaluate(xDual); // evaluate the function using AD
        gradient[i] = result.dual; // store derivative
        //printf("\nxDual[%d]: %f, grad[%d]: %f ",i,xDual[i].real,i,gradient[i]);
        xDual[i].dual = 0.0;
    }
}*/

__device__ double generate_random_double(unsigned int seed, double lower, double upper)
{ 
    hiprandState state;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state); // initialize cuRAND with unique sequence number
    return lower + (upper + (-lower)) * hiprand_uniform_double(&state);
    //return -5.0 + (-3.0 + 5.12) * hiprand_uniform_double(&state); // return scaled double
}

template<typename Function, int DIM>
__device__ double line_search(double f0, const double* x, const double* p, const double* g){
    const double c1=0.3;
    double alpha=1.0;
    double ddir = dot_product_device(g,p,DIM);
    double xTemp[DIM];
    for(int i=0;i<20;i++){
        for(int j=0;j<DIM;j++){
            xTemp[j] = x[j] + alpha*p[j];
        }
        double f1 = Function::evaluate(xTemp);
        if(f1 <= f0 + c1*alpha*ddir) break;
        alpha *= 0.5;
    }
    return alpha;
}

} // util namespace end

__device__ __forceinline__
double atomicMinDouble(double* addr, double val) {
    // reinterpret the address as 64‑bit unsigned
    unsigned long long* ptr = reinterpret_cast<unsigned long long*>(addr);
    unsigned long long old_bits = *ptr, assumed_bits;

    do {
        assumed_bits = old_bits;
        double old_val = __longlong_as_double(assumed_bits);
        // if the current value is already <= our candidate, nothing to do
        if (old_val <= val) break;
        // else try to swap in the new min value’s bit‐pattern
        unsigned long long new_bits = __double_as_longlong(val);
        old_bits = atomicCAS(ptr, assumed_bits, new_bits);
    } while (assumed_bits != old_bits);

    // return the previous minimum
    return __longlong_as_double(old_bits);
}

// kernel #1: initialize X, V, pBest; atomically seed gBestVal/gBestX
template<typename Function, int DIM>
__global__ void psoInitKernel(
    Function           func,
    double             lower,
    double             upper,
    double*            X,
    double*            V,
    double*            pBestX,
    double*            pBestVal,
    double*            gBestX,
    double*            gBestVal,
    int                N,
    uint64_t	       seed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    const double vel_range = (upper - lower) * 0.1;
    // const unsigned int seed = 1234u;
    //uint64_t counter = seed ^ (uint64_t)tid;
    unsigned int basePos = 1234u ^ (i * 0x9e3779b9);
    unsigned int baseVel = 2468u ^ (i * 0x7f4a7c15);
    // init position & velocity
    for (int d = 0; d < DIM; ++d) {
        unsigned int seedX = basePos ^ (d * 0x85ebca6bu);
        unsigned int seedV = baseVel ^ (d * 0xc2b2ae35u);
        double rx = util::generate_random_double(seedX, lower, upper);
	double rv = util::generate_random_double(seedV, -vel_range, vel_range);
        
	X[i*DIM + d]      = rx;
        V[i*DIM + d]      = rv;
        pBestX[i*DIM + d] = rx;
    }

    // eval personal best
    double fval = Function::evaluate(&X[i*DIM]);
    pBestVal[i] = fval;
    
    // atomic update of global best
    double oldGB = atomicMinDouble(gBestVal, fval);
    if (fval < oldGB) {
        // we’re the new champion: copy pBestX into gBestX
        for (int d = 0; d < DIM; ++d)
            gBestX[d] = pBestX[i*DIM + d];
    }
}

// kernel #2: one PSO iteration (velocity+position update, personal & global best)
template<typename Function, int DIM>
__global__ void psoIterKernel(
    Function           func,
    double             lower,
    double             upper,
    double             w, // weight inertia
    double             c1, // cognitive coefficient
    double             c2, // social coefficient
    double*            X,
    double*            V,
    double*            pBestX,
    double*            pBestVal,
    double*            gBestX,
    double*            gBestVal,
    double*            traj,        // pass nullptr if not saving
    bool               saveTraj,
    int                N,
    int                iter,
    uint64_t	       seed)       // iteration index, for RNG diversification
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    // const unsigned int seedBase = 5678u + iter*2;
    
    unsigned int base = 5678u + iter*1315423911u;
    // update velocity & position
    for (int d = 0; d < DIM; ++d) {
        unsigned int seed1 = base ^ (i*d + 0);
        unsigned int seed2 = base ^ (i*d + 1);
        
	double r1 = util::generate_random_double(seed1, 0.0, 1.0);
        double r2 = util::generate_random_double(seed2, 0.0, 1.0);
    	
	double xi = X[i*DIM + d];
        double vi = V[i*DIM + d];
        double pb = pBestX[i*DIM + d];
        double gb = gBestX[d];

        double nv = w*vi
                  + c1*r1*(pb - xi) // “cognitive” pull toward personal best
                  + c2*r2*(gb - xi); // “social” pull toward global best
        double nx = xi + nv;

        V[i*DIM + d] = nv;
        X[i*DIM + d] = nx;

        if (saveTraj) {
            // traj is laid out [iter][i][d]
            size_t idx = size_t(iter)*N*DIM + i*DIM + d;
            traj[idx] = nx;
        }
    }

    // evaluate at new position
    double fval = Function::evaluate(&X[i*DIM]);

    // personal best? no atomic needed, it's a private best position
    if (fval < pBestVal[i]) {
        pBestVal[i] = fval;
        for (int d = 0; d < DIM; ++d)
            pBestX[i*DIM + d] = X[i*DIM + d];
    }

    // global best?
    double oldGB = atomicMinDouble(gBestVal, fval);
    if (fval < oldGB) {
        for (int d = 0; d < DIM; ++d)
            gBestX[d] = X[i*DIM + d];
    }
    /*printf("it %d gBestVal = %.6f  at gBestX = [",i,fval);
    for (int d = 0; d < DIM; ++d)
        printf(" %8.4f", gBestX[d]);
    printf(" ]\n");*/
}

template<int DIM>
struct Result {
    int idx;
    int status; // 1 if converged, else if stopped_bc_someone_flipped_the_flag: 2, else 0
    double fval; // function value
    double gradientNorm;
    double coordinates[DIM];
    int iter;
};

__device__ int d_stopFlag;  // 0 = keep going; 1 = stop immediately
__device__ int d_convergedCount; // how many threads have converged?
__device__ int d_threadsRemaining;

template<typename Function, int DIM, unsigned int blockSize>
__global__ void optimizeKernel(const double lower,const double upper,
		const double* __restrict__ pso_array, // pso initialized positions
		double* deviceResults, double* deviceTrajectory, int N,const int MAX_ITER,const int requiredConverged,const double tolerance, Result<DIM>* result, bool save_trajectories = false) {
    extern __device__ int d_stopFlag;
    extern __device__ int d_threadsRemaining;
    extern __device__ int d_convergedCount;
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    //int early_stopping = 0;
    double H[DIM * DIM];
    double g[DIM], x[DIM], x_new[DIM], p[DIM], g_new[DIM], delta_x[DIM], delta_g[DIM];//, new_direction[DIM];
    //double tolerance = 1e-5;
    // Line Search params

    Result<DIM> r;
    r.status       = -1;     // assume “not converged” by default
    r.fval         = 333777.0;
    r.gradientNorm = 69.0;
    for (int d = 0; d < DIM; ++d) {
        r.coordinates[d] = 0.0;
    }
    r.iter = -1;
    r.idx = idx;
    util::initialize_identity_matrix(H, DIM);
    
    int num_steps = 0;

    // initialize x either from PSO array or fallback by RNG
    if (pso_array) {
        #pragma unroll
        for (int d = 0; d < DIM; ++d) {
            x[d] = pso_array[idx*DIM + d];
            if(idx == 0)
                printf("x[%d]=%0.7f\n", d, x[d]);
        }
    } else {
        unsigned int seed = 456;
        #pragma unroll
        for (int d = 0; d < DIM; ++d) {
            //x[d] = util::statelessUniform(idx,d,1,lower, upper, seed);
	    x[d] = util::generate_random_double(seed + idx*DIM + d, lower, upper);
        }
    }	
    
    double f0 = Function::evaluate(x);//rosenbrock_device(x, DIM);
    deviceResults[idx] = f0;
    double bestVal = f0;
    if (idx == 0) {
       printf("\n\nf0 = %f", f0);
    }
    int iter;
    util::calculateGradientUsingAD<Function, DIM>(x, g);
    for (iter = 0; iter < MAX_ITER; ++iter) {
        // check if somebody already asked to stop
	if (atomicAdd(&d_stopFlag, 0) != 0) { // atomicAdd here just to get a strong read-barrier 
            // CUDA will fetch a coherent copy of the integer from global memory. 
	    // as soon as one thread writes 1 into d_stopFlag via atomicExch, 
	    // the next time any thread does atomicAdd(&d_stopFlag, 0) it’ll see 1 and break.	   
            //printf("thread %d get outta dodge cuz we converged...", idx);
            r.status = 2;
            r.iter = iter;
	    r.fval = Function::evaluate(x);
            for(int d=0;d<DIM;d++){r.coordinates[d] = x[d];}
            r.gradientNorm = util::calculate_gradient_norm<DIM>(g); 
            break;
            
        }
	num_steps++;

        util::compute_search_direction<DIM>(p, H, g); //p = -H * g        

	// use the alpha obtained from the line search
	double alpha = util::line_search<Function,DIM>(bestVal, x, p, g);
	if(alpha == 0.0) {
            printf("Alpha is zero, no movement in iteration=%d\n", iter);
            alpha = 1e-3; 
        }

	// update current point x by taking a step size of alpha in the direction p
	for (int i = 0; i < DIM; ++i) {
            x_new[i] = x[i] + alpha * p[i];
	    delta_x[i] = x_new[i] - x[i];
	}

        double fnew = Function::evaluate(x_new);	
        // get the new gradient g_new at x_new
	util::calculateGradientUsingAD<Function, DIM>(x_new, g_new);
	
	// calculate new delta_x and delta_g
        for (int i = 0; i < DIM; ++i) {
	    delta_g[i] = g_new[i] - g[i]; // difference in gradient at the new point vs old point
        }

	// calculate the the dot product between the change in x and change in gradient using new point
        double delta_dot = util::dot_product_device(delta_x, delta_g, DIM);

	// bfgs update on H
        util::bfgs_update<DIM>(H, delta_x, delta_g, delta_dot);
        // only update x and g for next iteration if the new minima is smaller than previous
	//double min = Function::evaluate(x_new);//rosenbrock_device(x_new, DIM);
        if (fnew < bestVal) {
	   bestVal = fnew;
	   for(int i=0; i<DIM; ++i) {
	      x[i] = x_new[i];
	      g[i] = g_new[i];
	   } 
	}
        // refactor? yes 
        double grad_norm = util::calculate_gradient_norm<DIM>(g);
        if (grad_norm < tolerance) {
            // atomically increment the converged counter
            int oldCount = atomicAdd(&d_convergedCount, 1);
            int newCount = oldCount + 1;
            double fcurr = Function::evaluate(x);
            //printf("\nconverged for %d at iter=%d); f = %.6f;",idx, iter,fcurr);
            //for (int d = 0; d < DIM; ++d) { printf(" % .6f", x[d]);}
            //printf(" ]\n");
            r.status       = 1;
            r.gradientNorm = grad_norm;
            r.fval         = Function::evaluate(x);
            r.iter         = iter;
            for (int d = 0; d < DIM; ++d) {
                r.coordinates[d] = x[d];
            }	    
	    // if we just hit the threshold set by the user, the VERY FIRST thread to do so
            // sets d_stopFlag=1 so everyone else exits on their next check
            if (newCount == requiredConverged) {
                // flip the global stop flag
                atomicExch(&d_stopFlag, 1);
                __threadfence();
                printf("\nThread %d is the %d%s converged thread (iter=%d); fn = %.6f.\n",idx, newCount,(newCount==1? "st" : newCount==2? "nd" : newCount==3? "rd" : "th"),iter,fcurr);
            }
            // in _any_ case, whether we were the last to converge or not, 
            // we are individually done so break
            break;
        }

	//  deviceTrajectory layout: idx * (MAX_ITER * DIM) + iter * DIM + i
	if (save_trajectories) {
	   for (int i = 0; i < DIM; i++) {
               deviceTrajectory[idx * (MAX_ITER * DIM) + iter * DIM + i] = x[i];
           }
	} 

	//for(int i=0; i<DIM; ++i) {x[i] = x_new[i];}
    }// end bfgs loop
    // if we broek out because we hit the max numberof iterations, then its a surrender
    if(MAX_ITER == iter) {
        r.status = 0; // surrender
        r.iter = iter;
        r.gradientNorm = util::calculate_gradient_norm<DIM>(g);
        r.fval = Function::evaluate(x);
        for (int d = 0; d < DIM; ++d) { r.coordinates[d] = x[d];}
    }
    deviceResults[idx] = Function::evaluate(x);
    result[idx] = r;
}// end optimizerKernel

bool askUser2saveTrajectories() {
    std::cout << "Save optimization trajectories? (y/n): ";
    char ans;
    std::cin >> ans;
    std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
    return (ans == 'y' || ans == 'Y');
}

void createOutputDirs(const std::string &path) {
    std::filesystem::create_directories(path);
}

hipError_t writeTrajectoryData(
    double* hostTrajectory,
    int N, int MAX_ITER, int DIM,
    const std::string &fun_name,
    const std::string &basePath
) {
    // construct the directory path and create it.
    std::string dirPath = basePath + "/" + fun_name + "/"
       + std::to_string(DIM) + "d/" + std::to_string(MAX_ITER * N) + "/trajectories";
    std::filesystem::create_directories(dirPath);
    //createOutputDirs(dirPath);

    // the final filename.
    std::string filename = dirPath + "/"
                         + std::to_string(MAX_ITER) + "it_"
                         + std::to_string(N) + ".tsv";

    std::ofstream stepOut(filename);
    stepOut << "OptIndex\tStep";
    for (int d = 0; d < DIM; d++)
        stepOut << "\tX_" << d;
    stepOut << "\n";
    stepOut << std::scientific << std::setprecision(17);
    for (int i = 0; i < N; i++) {
        for (int it = 0; it < MAX_ITER; it++) {
            stepOut << i << "\t" << it;
            for (int d = 0; d < DIM; d++) {
                stepOut << "\t" << hostTrajectory[i * (MAX_ITER * DIM) + it * DIM + d];
            }
	    stepOut << "\n";
        }
    }
    stepOut.close();
    return hipSuccess;
}


// make it write to std::cout + dump to file
template<int DIM>
void dump_data_2_file(const Result<DIM>* h_results,const std::string fun_name,const int N, const int PSO_ITER) {
    std::string filename = "./data/" + fun_name +"_" + std::to_string(PSO_ITER)+"psoit_" + std::to_string(DIM) + "d_particledata.tsv";

    bool file_exists = std::filesystem::exists(filename);
    bool file_empty = file_exists ? (std::filesystem::file_size(filename) == 0) : true;
    std::ofstream outfile(filename, std::ios::app);
    if (!outfile.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    // if file is new or empty, let us write the header
    if (file_empty) {
        outfile << "fun\tidx\tstatus\titer\tfval\tnorm";
        for (int i = 0; i < DIM; i++)
            outfile << "\tcoord_" << i;
        outfile << std::endl;
    }// end if file is empty

    std::string tab = "\t";
    int countConverged = 0, surrender = 0, stopped = 0;
    for (int i = 0; i < N; ++i) {
        outfile << fun_name << tab << i << tab << std::scientific; 
        if (h_results[i].status == 1) {
            countConverged++;
            outfile << 1 << tab;
        } else if(h_results[i].status == 2) { // particle was stopped early
            stopped++;
            outfile << 2 << tab;
            //printf("Thread %d was stopped early (iter=%d)\n", i, h_results[i].iter);
        } else {
            surrender++;
            outfile << 0 << tab;
        }
        outfile << h_results[i].iter << tab << h_results[i].fval << tab << h_results[i].gradientNorm;
        for(int d = 0; d < DIM; ++d) {
            outfile << "\t"<< h_results[i].coordinates[d];
        }
        outfile << std::endl;
    }
    std::cout << "\ndumped data 2 "<< filename << "\n"<<countConverged <<" converged, "<<stopped << " stopped early, "<<surrender<<" surrendered\n"; 
    //printf("\ndumped data 2 %s\n%d converged, %d stopped early, %d surrendered\n",filename.c_str(),countConverged, stopped, surrender);
}


void append_results_2_tsv(const int dim,const int N, const std::string fun_name,float ms_init, float ms_pso,float ms_opt,const int max_iter, const int pso_iter,const double error,const double globalMin, double* hostCoordinates, const int idx, const int status, const double norm) {
        std::string filename = "zeus_" + std::to_string(dim) + "d_results.tsv";
        std::ofstream outfile(filename, std::ios::app);
        
        bool file_exists = std::filesystem::exists(filename);
        bool file_empty = file_exists ? (std::filesystem::file_size(filename) == 0) : true;
        //std::ofstream outfile(filename, std::ios::app);
        if (!outfile.is_open()) {
            std::cerr << "Error opening file: " << filename << std::endl;
            return;
        }

        // if file is new or empty, let us write the header
        if (file_empty) {
            outfile << "fun\tN\tidx\tstatus\tbfgs_iter\tpso_iter\ttime\terror\tfval\tnorm\t";
            for (int i = 0; i < dim; i++)
                outfile << "\tcoord_" << i;
            outfile << std::endl;
        }// end if file is empty
        
        double time_seconds = std::numeric_limits<double>::infinity();
        if (pso_iter > 0) {
            time_seconds = (ms_init+ms_pso+ms_opt);
            printf("total time = pso + bfgs = total time = %0.4f ms\n", time_seconds);
        } else {
            time_seconds = ms_opt;
            printf("bfgs time = total time = %.4f ms\n", time_seconds);
        }
        outfile << fun_name << "\t" << N << "\t"<<idx<<"\t"<<status <<"\t" << max_iter << "\t" << pso_iter << "\t"
            << time_seconds << "\t"
            << std::scientific << error << "\t" << globalMin << "\t" << norm <<"\t" ;
        for (int i = 0; i < dim; i++) {
            outfile << hostCoordinates[i];
            if (i < dim - 1)
                outfile << "\t";
        }
        outfile << "\n";
        outfile.close();
        printf("results are saved to %s", filename.c_str());
}// end append_results_2_tsv

template<typename Function, int DIM>
double* launch_pso(const int PSO_ITER,const int N,const double lower,const double upper, float& ms_init, float& ms_pso) {
        // allocate PSO buffers on device
        double *dX, *dV, *dPBestVal, *dGBestX, *dGBestVal, *dPBestX;
        hipMalloc(&dX,        N*DIM*sizeof(double));
        hipMalloc(&dV,        N*DIM*sizeof(double));
        hipMalloc(&dPBestX,   N*DIM*sizeof(double));
        hipMalloc(&dPBestVal, N   *sizeof(double));
        hipMalloc(&dGBestX,   DIM *sizeof(double));
        hipMalloc(&dGBestVal, sizeof(double));
        int zero = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(d_stopFlag), &zero, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(d_threadsRemaining), &N, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(d_convergedCount),   &zero, sizeof(int));
        // set seed to infinity
        {
            double inf = std::numeric_limits<double>::infinity();
            hipMemcpy(dGBestVal, &inf, sizeof(inf), hipMemcpyHostToDevice);
        }

        dim3 psoBlock(256);
        dim3 psoGrid((N + psoBlock.x - 1) / psoBlock.x);

        // host-side buffers for printing
        double hostGBestVal;
        std::vector<double> hostGBestX(DIM);

        // PSO‐init Kernel
        hipEvent_t t0, t1;
        hipEventCreate(&t0);
        hipEventCreate(&t1);
        hipEventRecord(t0);
        psoInitKernel<Function,DIM><<<psoGrid,psoBlock>>>(
             Function(), lower, upper,
             dX, dV,
             dPBestX, dPBestVal,
             dGBestX, dGBestVal,
             N,1234567);
        hipDeviceSynchronize();
        hipEventRecord(t1);
        hipEventSynchronize(t1);

        hipEventElapsedTime(&ms_init, t0, t1);
        printf("PSO‑Init Kernel execution time = %.4f ms\n", ms_init);

        // copy back and print initial global best
        hipMemcpy(&hostGBestVal, dGBestVal, sizeof(double),        hipMemcpyDeviceToHost);
        hipMemcpy(hostGBestX.data(),  dGBestX,   DIM*sizeof(double), hipMemcpyDeviceToHost);

        printf("Initial PSO gBestVal = %.6e at gBestX = [", hostGBestVal);
        for(int d=0; d<DIM; ++d) printf(" %.4f", hostGBestX[d]);
        printf(" ]\n\n");

        // PSO iterations (each timed + host‐print) ––
        const double w  = 0.5, c1 = 1.2, c2 = 1.5;
        for(int iter=1; iter<PSO_ITER+1; ++iter) {
            hipEventRecord(t0);
            psoIterKernel<Function,DIM><<<psoGrid,psoBlock>>>(
                Function(),
                lower, upper,
                w, c1, c2,
                dX, dV,
                dPBestX, dPBestVal,
                dGBestX, dGBestVal,
                nullptr,// traj
                false,//saveTraj
                N, iter, 1234567);
            hipDeviceSynchronize();
            hipEventRecord(t1);
            hipEventSynchronize(t1);

            float ms_iter=0;
            hipEventElapsedTime(&ms_iter, t0, t1);
            hipMemcpy(&hostGBestVal, dGBestVal, sizeof(double),        hipMemcpyDeviceToHost);
            hipMemcpy(hostGBestX.data(),  dGBestX,   DIM*sizeof(double), hipMemcpyDeviceToHost);

            printf("PSO‑Iter %2d execution time = %.3f ms   gBestVal = %.6e at [",iter, ms_iter, hostGBestVal);
            for(int d=0; d<DIM; ++d) printf(" %.4f", hostGBestX[d]);
            printf(" ]\n");
            ms_pso += ms_iter;
        }// end pso loop
        printf("total pso time = %.3f\n", ms_pso+ms_init);
        /*double *hPBestX = new double[DIM]; 
        hipMemcpy(hPBestX,
           dPBestX,
           DIM * sizeof(double),
           hipMemcpyDeviceToHost); 
        */
    hipEventDestroy(t0);
    hipEventDestroy(t1);
    hipFree(dX);
    hipFree(dV);
    //hipFree(dPBestX);
    hipFree(dPBestVal);
    hipFree(dGBestX);
    hipFree(dGBestVal);
    return dPBestX;
}

template<int DIM>
Result<DIM> launch_reduction(int N, double* deviceResults,Result<DIM>* h_results) {
    // ArgMin & final print
    hipcub::KeyValuePair<int,double>* deviceArgMin;
    hipMalloc(&deviceArgMin,     sizeof(*deviceArgMin));
    void*  d_temp_storage = nullptr;
    size_t temp_bytes      = 0;
    hipcub::DeviceReduce::ArgMin(
        d_temp_storage, temp_bytes,
        deviceResults, deviceArgMin, N);
    hipMalloc(&d_temp_storage, temp_bytes);
    hipcub::DeviceReduce::ArgMin(
        d_temp_storage, temp_bytes,
        deviceResults, deviceArgMin, N);

    hipcub::KeyValuePair<int,double> h_argMin;
    hipMemcpy(&h_argMin, deviceArgMin,
               sizeof(h_argMin),
               hipMemcpyDeviceToHost);

    int    globalMinIndex = h_argMin.key;
    //double globalMin      = h_argMin.value;

    // copy back the entire array of Result structs:
    //Result* h_results = new Result[N];
    //hipMemcpy(h_results, d_results,N * sizeof(Result),hipMemcpyDeviceToHost);
    //Result<DIM>* hn_results = new Result<DIM>[N];
    //hipMemcpy(h_results,d_results,N * sizeof(Result<DIM>),hipMemcpyDeviceToHost);
    // print the “best” thread’s full record
    Result best = h_results[globalMinIndex];
    printf("Global best summary:\n");
    printf("   idx          = %d\n", best.idx);
    printf("   status       = %d\n", best.status);
    printf("   fval         = %.6f\n",best.fval);
    printf("   gradientNorm = %.6f\n",best.gradientNorm);
    printf("   iter         = %d\n",best.iter);
    printf("   coords       = [");
    for (int d = 0; d < DIM; ++d) {
         printf(" %.7f", best.coordinates[d]);
    }
    printf(" ]\n");

    hipFree(deviceResults);
    hipFree(deviceArgMin);
    hipFree(d_temp_storage);
    return best;
}

template<typename Function, int DIM>
Result<DIM> launch_bfgs(const int N,const int pso_iter, const int MAX_ITER, const double upper, const double lower,double* pso_results_device,double* hostResults, double* deviceTrajectory, const int requiredConverged, const double tolerance, bool save_trajectories, float& ms_opt, std::string fun_name) {
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize, &blockSize,
        optimizeKernel<Function,DIM,128>,
        0, N);
    printf("\nRecommended block size: %d\n", blockSize);
    
    // prepare optimizer buffers & copy hostResults --> device
    double* deviceResults;
    hipMalloc(&deviceResults,    N * sizeof(double));
    hipMemcpy(deviceResults, hostResults, N*sizeof(double), hipMemcpyHostToDevice);

    dim3 optBlock(blockSize);
    dim3 optGrid((N + blockSize - 1) / blockSize);

    // optimizeKernel time
    hipEvent_t startOpt, stopOpt;
    hipEventCreate(&startOpt);
    hipEventCreate(&stopOpt);
    hipEventRecord(startOpt);

    Result<DIM>* h_results = new Result<DIM>[N];            // host copy
    Result<DIM>* d_results = nullptr;
    hipMalloc(&d_results, N * sizeof(Result<DIM>));
    /*
    for(int i=0;i<DIM;i++){ 
        std::cout << hPBestX[i] << " ";
    }*/
    std::cout << std::endl;
    if (save_trajectories) {
        hipMalloc(&deviceTrajectory, N*MAX_ITER*DIM*sizeof(double));
        optimizeKernel<Function,DIM,128>
            <<<optGrid,optBlock>>>(
                lower, upper,
                pso_results_device,
                deviceResults,
                deviceTrajectory,
                N,MAX_ITER,requiredConverged,tolerance,d_results,
                /*saveTraj=*/true);
    } else {
        optimizeKernel<Function,DIM,128>
            <<<optGrid,optBlock>>>(
                lower, upper,
                pso_results_device,
                deviceResults,
                /*traj=*/nullptr,
                N,MAX_ITER,requiredConverged,tolerance,d_results);
    }
    hipDeviceSynchronize();
    hipEventRecord(stopOpt);
    hipEventSynchronize(stopOpt);
    hipEventElapsedTime(&ms_opt, startOpt, stopOpt);
    printf("\nOptimization Kernel execution time = %.3f ms\n", ms_opt);
    hipEventDestroy(startOpt);
    hipEventDestroy(stopOpt);

    hipMemcpy(h_results, d_results, N * sizeof(Result<DIM>), hipMemcpyDeviceToHost);

    dump_data_2_file(h_results, fun_name, N, pso_iter);
    /*int countConverged = 0, surrender = 0, stopped = 0;
    for (int i = 0; i < N; ++i) {
        if (h_results[i].status == 1) { 
            countConverged++;
        } else if(h_results[i].status == 2) { // particle was stopped early
            stopped++;
        } else {
            surrender++;
        }
    }
    printf("\n%d converged, %d stopped early, %d surrendered\n",countConverged, stopped, surrender);
    */
    Result best = launch_reduction<DIM>(N, deviceResults, h_results);
    
    return best;
}

double calculate_euclidean_error(const std::string fun_name, const double* coordinates, const int dim) {
   double sum_sq = 0.0;
   if(fun_name == "rosenbrock") {
      for(int i=0;i<dim;i++) {
         double diff = coordinates[i] - 1.0;
         sum_sq += diff * diff;
      }
   } else if(fun_name == "rastrigin" || fun_name == "ackley") { // both rastrigin and ackley have the same coordinates for the global minimum
      for (int i = 0; i < dim; ++i) {
         sum_sq += coordinates[i] * coordinates[i];
      }
   }
   return std::sqrt(sum_sq);
}// end calculate_euclidean_error

template<typename Function, int DIM>
Result<DIM> Zeus(const double lower,const double upper, double* hostResults,int N,int MAX_ITER, int PSO_ITER, int requiredConverged,std::string fun_name, double tolerance)
{
    int blockSize, minGridSize;
    hipOccupancyMaxPotentialBlockSize(
        &minGridSize, &blockSize,
        optimizeKernel<Function,DIM,128>,
        0, N);
    printf("Recommended block size: %d\n", blockSize);

    bool save_trajectories = askUser2saveTrajectories();
    double* deviceTrajectory = nullptr;
    double* pso_results_device=nullptr;
    float ms_init = 0.0f, ms_pso = 0.0f; 
    if(PSO_ITER > 0) {
        pso_results_device = launch_pso<Function, DIM>(PSO_ITER, N,lower, upper, ms_init,ms_pso);
        printf("pso init: %.2f main loop: %.2f", ms_init, ms_pso); 
    }// end if pso_iter > 0 
    if(!pso_results_device) 
       std::cout <<"still null" << std::endl;
    float ms_opt = 0.0f;
    Result best = launch_bfgs<Function, DIM>(N,PSO_ITER, MAX_ITER,upper, lower, pso_results_device, hostResults, deviceTrajectory, requiredConverged,tolerance, save_trajectories, ms_opt, fun_name);
    if(PSO_ITER > 0) { // optimzation routine is finished, so we can free that array on the device
         hipFree(pso_results_device);
    } 

    double error = calculate_euclidean_error(fun_name, best.coordinates, DIM);
    append_results_2_tsv(DIM,N,fun_name,ms_init,ms_pso,ms_opt,MAX_ITER, PSO_ITER,error,best.fval, best.coordinates, best.idx, best.status, best.gradientNorm);
     
    hipError_t cuda_error  = hipGetLastError();
    if (cuda_error != hipSuccess) { 
        printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
    } else {
        printf("\nSuccess!! No Error!\n");
    }
    return best;
}// end Zeus

template<typename Function, int DIM>
void runOptimizationKernel(double lower, double upper, double* hostResults, int N, int MAX_ITER,int PSO_ITERS,int requiredConverged, std::string fun_name, double tolerance) {
//void runOptimizationKernel(double* hostResults, int N, int dim) {
    /*printf("first 20 hostResults\n");
    for(int i=0;i<20;i++) {
       printf(" %f ",hostResults[i]);
    }
    printf("\n");
    */
    Result best = Zeus<Function, DIM>(lower, upper, hostResults, N, MAX_ITER, PSO_ITERS, requiredConverged, fun_name,tolerance);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //printf("Sorting the array with %d elements... ", N);  
    hipEventRecord(start);
    //quickSort(hostResults, 0, N - 1);
    hipEventRecord(stop);
    float milli = 0;
    hipEventElapsedTime(&milli, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //printf("took %f ms\n",  milli);    

    /*printf("first 20 function values in hostResults\n");
    for(int i=0;i<20;i++) {
       printf(" %f ",hostResults[i]);
    }*/
    printf("\n");
//hipMemGetInfo
}


template<int dim>
void selectAndRunOptimization(double lower, double upper,double* hostResults, int N, int MAX_ITER,int PSO_ITERS,int requiredConverged, double tolerance) {
    int choice;
    std::cout << "\nSelect function to optimize:\n"
              << " 1. Rosenbrock\n"
              << " 2. Rastrigin\n"
              << " 3. Ackley\n";
    // Only show 2D-only options when dim == 2.
    if constexpr (dim == 2) {
        std::cout << " 4. GoldsteinPrice\n"
                  << " 5. Eggholder\n"
                  << " 6. Himmelblau\n";
    }
    std::cout << " 7. Custom (user-defined objective via expression or kernel file)\n"
              << "Choice: ";
    std::cin >> choice;
    std::cin.ignore();

    switch(choice) {
        case 1:
            std::cout << "\n\n\tRosenbrock Function\n" << std::endl;
            runOptimizationKernel<util::Rosenbrock<dim>, dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged,"rosenbrock", tolerance);
            break;
        case 2:
            std::cout << "\n\n\tRastrigin Function\n" << std::endl;
            runOptimizationKernel<util::Rastrigin<dim>, dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged, "rastrigin", tolerance);
            break;
        case 3:
            std::cout << "\n\n\tAckley Function\n" << std::endl;
            runOptimizationKernel<util::Ackley<dim>, dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged, "ackley", tolerance);
            break;
        case 4:
            if constexpr (dim != 2) {
                std::cerr << "Error: GoldsteinPrice is defined for 2 dimensions only.\n";
            } else {
                std::cout << "\n\n\tGoldsteinPrice Function\n" << std::endl;
                runOptimizationKernel<util::GoldsteinPrice<dim>, dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged, "goldstein", tolerance);
            }
            break;
        case 5:
            if constexpr (dim != 2) {
                std::cerr << "Error: Eggholder is defined for 2 dimensions only.\n";
            } else {
                std::cout << "\n\n\tEggholder Function\n" << std::endl;
                runOptimizationKernel<util::Eggholder<dim>, dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged, "eggholder", tolerance);
            }
            break;
        case 6:
            if constexpr (dim != 2) {
                std::cerr << "Error: Himmelblau is defined for 2 dimensions only.\n";
            } else {
                std::cout << "\n\n\tHimmelblau Function\n" << std::endl;
                runOptimizationKernel<util::Himmelblau<dim>, dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged, "himmelblau", tolerance);
            }
            break;
        case 7:
            std::cout << "\n\n\tCustom User-Defined Function\n" << std::endl;
            // for a more complex custom function, one option is to let the user provide a path
            // to a cuda file and compile it at runtime. 
            //runOptimizationKernel<UserDefined<dim>, dim>(lower, upper, hostResults, hostIndices,
            //                                             hostCoordinates, N, MAX_ITER);
            break;
        default:
            std::cerr << "Invalid selection!\n";
            exit(1);
    }
}

#ifndef UNIT_TEST
int main(int argc, char* argv[]) {
    printf("Production main() running\n");
    if (argc != 8) {
	 std::cerr << "Usage: " << argv[0] << " <lower_bound> <upper_bound> <max_iter> <pso_iters> <converged> <number_of_optimizations> <tolerance>\n";
        return 1;
    }
    double lower = std::atof(argv[1]);
    double upper = std::atof(argv[2]);   	
    int MAX_ITER = std::stoi(argv[3]);
    int PSO_ITERS = std::stoi(argv[4]);
    int requiredConverged = std::stoi(argv[5]);
    int N = std::stoi(argv[6]);
    double tolerance = std::stod(argv[7]);
    std::cout << "Tolerance: " << std::setprecision(10) << tolerance << "\n";

    //const size_t N = 128*4;//1024*128*16;//pow(10,5.5);//128*1024*3;//*1024*128;
    const int dim = 10;
    double hostResults[N];// = new double[N];
    std::cout << "number of optimizations = " << N << " max_iter = " << MAX_ITER << " dim = " << dim << std::endl;
     
    double f0 = 333777; // initial function value

    // logic to set the stact size limit to 65 kB per thread 
    size_t currentStackSize = 0;
    hipDeviceGetLimit(&currentStackSize, hipLimitStackSize);
    printf("Current stack size: %zu bytes\n", currentStackSize);
    size_t newStackSize = 64  * 1024; // 65 kB
    hipError_t err = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
    if (err != hipSuccess) {
        printf("hipDeviceSetLimit error: %s\n", hipGetErrorString(err));
        return 1;
    } else {
            printf("Successfully set stack size to %zu bytes\n", newStackSize);
    }// end stack size limit

    char cont = 'y';
    while (cont == 'y' || cont == 'Y') {
        for (int i = 0; i < N; i++) {
            hostResults[i] = f0;
        }
        selectAndRunOptimization<dim>(lower, upper, hostResults, N, MAX_ITER,PSO_ITERS, requiredConverged, tolerance);
        std::cout << "\nDo you want to optimize another function? (y/n): ";
        std::cin >> cont;
        std::cin.ignore();
    }
    

    //for(int i=0; i<N; i++) {
    //    hostResults[i] = f0;
    //}
    //selectAndRunOptimization<dim>(lower, upper, hostResults, hostIndices, hostCoordinates, N, MAX_ITER);
    return 0;
}
#endif
