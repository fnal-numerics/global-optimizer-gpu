#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>
#include <limits>
#include <stdexcept>
#include <fstream>
#include <math.h>
#include <iomanip>
#include <cstdlib>
#include <random>
#include <algorithm>
#include <cstring>

#include <filesystem>

// #include "fastPRNG.h"

// #include <Random123/philox.h>             // first, so mulhilo32 is chosen
// only once #include <Random123/uniform.hpp>

// #include "random123/philox_uniform.cuh"
// using fastPRNG;
// fastXS64 fastR;

namespace dual {

  class DualNumber {
  public:
    double real;
    double dual;

    __host__ __device__
    DualNumber(double real = 0.0, double dual = 0.0)
      : real(real), dual(dual)
    {}

    __host__ __device__ DualNumber&
    operator+=(const DualNumber& rhs)
    {
      real += rhs.real;
      dual += rhs.dual;
      return *this;
    }

    __host__ __device__ DualNumber
    operator+(const DualNumber& rhs) const
    {
      return DualNumber(real + rhs.real, dual + rhs.dual);
    }

    __host__ __device__ DualNumber
    operator-(const DualNumber& rhs) const
    {
      return DualNumber(real - rhs.real, dual - rhs.dual);
    }

    __host__ __device__ DualNumber
    operator*(const DualNumber& rhs) const
    {
      return DualNumber(real * rhs.real, dual * rhs.real + real * rhs.dual);
    }

    __host__ __device__ DualNumber
    operator/(const DualNumber& rhs) const
    {
      double denom = rhs.real * rhs.real;
      return DualNumber(real / rhs.real,
                        (dual * rhs.real - real * rhs.dual) / denom);
    }
    // operator for double - DualNumber
    __host__ __device__ friend DualNumber
    operator-(double lhs, const DualNumber& rhs)
    {
      return DualNumber(lhs - rhs.real, -rhs.dual);
    }

    // operator for double * DualNumber
    __host__ __device__ friend DualNumber
    operator*(double lhs, const DualNumber& rhs)
    {
      return DualNumber(lhs * rhs.real, lhs * rhs.dual);
    }
  };

  __host__ __device__ inline dual::DualNumber
  dual_abs(const dual::DualNumber& a)
  {
    return (a.real < 0.0) ? dual::DualNumber(-a.real, -a.dual) : a;
  }

  __host__ __device__ DualNumber
  sin(const DualNumber& x)
  {
    return DualNumber(sinf(x.real), x.dual * cosf(x.real));
  }

  __host__ __device__ DualNumber
  cos(const DualNumber& x)
  {
    return DualNumber(cosf(x.real), -x.dual * sinf(x.real));
  }

  __host__ __device__ DualNumber
  exp(const DualNumber& x)
  {
    double ex = expf(x.real);
    return DualNumber(ex, x.dual * ex);
  }

  __host__ __device__ DualNumber
  sqrt(const DualNumber& x)
  {
    double sr = sqrtf(x.real);
    return DualNumber(sr, x.dual / (2.0 * sr));
  }

  __host__ __device__ DualNumber
  atan2(const DualNumber& y, const DualNumber& x)
  {
    double denom = x.real * x.real + y.real * y.real;
    return DualNumber(atan2f(y.real, x.real),
                      (x.real * y.dual - y.real * x.dual) / denom);
  }

  template <typename T>
  __host__ __device__ T
  pow(const T& base, double exponent)
  {
    return T(powf(base.real, exponent),
             exponent * powf(base.real, exponent - 1) * base.dual);
  }

} // end of dual

namespace util {

  __device__ double
  dot_product_device(const double* a, const double* b, int size)
  {
    double sum = 0.0;
    for (int i = 0; i < size; ++i) {
      sum += a[i] * b[i];
    }
    return sum;
  }

  __device__ void
  outer_product_device(const double* v1,
                       const double* v2,
                       double* result,
                       int size)
  {
    for (int i = 0; i < size; ++i) {
      for (int j = 0; j < size; ++j) {
        int idx = i * size + j;
        if (idx < size * size) {
          result[idx] = v1[i] * v2[j];
        } else {
          printf("outer product out of bounds..\ndim=%d i*size+j=%d\n",
                 size,
                 i * size + j);
        }
      }
    }
  }

  // wrap kernel definitions extern "C" block so that their symbols are exported
  // with C linkage
  extern "C" {
  __device__ __noinline__ void
  vector_add(const double* a, const double* b, double* result, int size)
  {
    for (int i = 0; i < size; ++i) {
      result[i] = a[i] + b[i];
    }
  }

  __device__ __noinline__ void
  vector_scale(const double* a, double scalar, double* result, int dim)
  {
    for (int i = 0; i < dim; ++i) {
      result[i] = a[i] * scalar;
    }
  }
  } // end extern C

  __device__ void
  initialize_identity_matrix(double* H, int dim)
  {
    for (int i = 0; i < dim; ++i) {
      for (int j = 0; j < dim; ++j) {
        H[i * dim + j] = (i == j) ? 1.0 : 0.0;
      }
    }
  }

  __device__ bool
  valid(double x)
  {
    if (isinf(x)) {
      return false;
    } else if (isnan(x)) {
      return false;
    } else {
      return true;
    }
  }

  __device__ double
  pow2(double x)
  {
    return x * x;
  }

  __device__ void
  initialize_identity_matrix_device(double* H, int n)
  {
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n; ++j) {
        H[i * n + j] = (i == j) ? 1.0 : 0.0;
      }
    }
  }

  template <int DIM>
  __device__ void
  matrix_multiply_device(const double* A, const double* B, double* C)
  {
    for (int i = 0; i < DIM; ++i) {
      for (int j = 0; j < DIM; ++j) {
        double sum = 0.0;
        for (int k = 0; k < DIM; ++k) {
          sum += A[i * DIM + k] * B[k * DIM + j];
        }
        C[i * DIM + j] = sum;
      }
    }
  }

  // BFGS update with compile-time dimension
  template <int DIM>
  __device__ void
  bfgs_update(double* H, const double* s, const double* y, double sTy)
  {
    if (::fabs(sTy) < 1e-14)
      return;
    double rho = 1.0 / sTy;

    // Compute H_new element-wise without allocating large temporary matrices.
    // H_new = (I - rho * s * y^T) * H * (I - rho * y * s^T) + rho * s * s^T
    double H_new[DIM * DIM]; // Temporary array (DIM^2 elements)

    for (int i = 0; i < DIM; i++) {
      for (int j = 0; j < DIM; j++) {
        double sum = 0.0;
        for (int k = 0; k < DIM; k++) {
          // Compute element (i,k) of (I - rho * s * y^T)
          double A_ik = ((i == k) ? 1.0 : 0.0) - rho * s[i] * y[k];
          double inner = 0.0;
          for (int m = 0; m < DIM; m++) {
            // Compute element (m,j) of (I - rho * y * s^T)
            double B_mj = ((m == j) ? 1.0 : 0.0) - rho * y[m] * s[j];
            inner += H[k * DIM + m] * B_mj;
          }
          sum += A_ik * inner;
        }
        // Add the rho * s * s^T term
        H_new[i * DIM + j] = sum + rho * s[i] * s[j];
      }
    }

    // Copy H_new back into H
    for (int i = 0; i < DIM * DIM; i++) {
      H[i] = H_new[i];
    }
  }

  // function to calculate scalar directional direvative d = g * p
  __device__ double
  directional_derivative(const double* grad, const double* p, int dim)
  {
    double d = 0.0;
    for (int i = 0; i < dim; ++i) {
      d += grad[i] * p[i];
    }
    return d;
  }

  template <int DIM>
  __device__ dual::DualNumber
  rosenbrock(const dual::DualNumber* x)
  {
    dual::DualNumber sum = 0.0;
    for (int i = 0; i < DIM - 1; ++i) {
      sum += dual::pow(1 - x[i], 2) +
             100 * dual::pow(x[i + 1] - dual::pow(x[i], 2), 2);
      // sum = sum + 100 * (x[i + 1] - x[i] * x[i]) * (x[i + 1] - x[i] * x[i]) +
      // (1 - x[i]) * (1 - x[i]);
    }
    return sum;
  }

  template <int DIM>
  __host__ __device__ double
  rosenbrock(const double* x)
  {
    double sum = 0.0;
    for (int i = 0; i < DIM - 1; ++i) {
      sum += 100 * (x[i + 1] - x[i] * x[i]) * (x[i + 1] - x[i] * x[i]) +
             (1 - x[i]) * (1 - x[i]);
    }
    return sum;
  }

  template <int DIM>
  __device__ dual::DualNumber
  rastrigin(const dual::DualNumber* x)
  {
    const double A = 10.0;
    dual::DualNumber sum(A * DIM, 0.0);
    for (int i = 0; i < DIM; i++) {
      dual::DualNumber xi_sq = x[i] * x[i];
      dual::DualNumber cterm = cos(2.0 * M_PI * x[i]);
      sum = sum + (xi_sq - A * cterm);
    }
    return sum;
  }

  template <int DIM>
  __host__ __device__ double
  rastrigin(const double* x)
  {
    const double A = 10.0;
    double val = A * DIM;
    for (int i = 0; i < DIM; i++) {
      double xi = x[i];
      val += xi * xi - A * ::cosf(2.0f * M_PI * xi);
    }
    return val;
    /*
    	double sum = 10 * DIM;
    for (int i = 0; i < DIM; ++i) {
        sum += x[i] * x[i] - 10 * cos(2 * M_PI * x[i]);
    }
    return sum;
*/}

    template <int DIM>
    struct Rosenbrock {
      __device__ static dual::DualNumber
      evaluate(const dual::DualNumber* x)
      {
        // printf("in dualrosen");
        return rosenbrock<DIM>(x);
      }

      __host__ __device__ static double
      evaluate(const double* x)
      {
        return rosenbrock<DIM>(x);
      }
    };

    template <int DIM>
    struct Rastrigin {
      __device__ static dual::DualNumber
      evaluate(const dual::DualNumber* x)
      {
        return rastrigin<DIM>(x);
      }

      __host__ __device__ static double
      evaluate(const double* x)
      {
        return rastrigin<DIM>(x);
      }
    };

    // Ackley Function (general d-dimensions)
    //   f(x) = -20 exp\Bigl(-0.2\sqrt{\frac{1}{d}\sum_{i=1}^{d}x_i^2}\Bigr)
    //          - exp\Bigl(\frac{1}{d}\sum_{i=1}^{d}\cos(2\pi x_i)\Bigr)
    //          + 20 + e
    template <int DIM>
    __device__ dual::DualNumber
    ackley(const dual::DualNumber* x)
    {
      dual::DualNumber sum_sq = 0.0;
      dual::DualNumber sum_cos = 0.0;
      for (int i = 0; i < DIM; ++i) {
        sum_sq += dual::pow(x[i], 2);
        sum_cos += dual::cos(2.0 * M_PI * x[i]);
      }
      dual::DualNumber term1 =
        dual::DualNumber(-20.0) * dual::exp(-0.2 * dual::sqrt(sum_sq / DIM));
      dual::DualNumber term2 = dual::DualNumber(0.0) - dual::exp(sum_cos / DIM);
      return term1 + term2 + 20.0 + dual::exp(1.0);
    }

    template <int DIM>
    __host__ __device__ double
    ackley(const double* x)
    {
      double sum_sq = 0.0;
      double sum_cos = 0.0;
      for (int i = 0; i < DIM; ++i) {
        sum_sq += x[i] * x[i];
        sum_cos += cos(2.0 * M_PI * x[i]);
      }
      double term1 = -20.0 * exp(-0.2 * sqrt(sum_sq / DIM));
      double term2 = -exp(sum_cos / DIM);
      return term1 + term2 + 20.0 + exp(1.0);
    }

    template <int DIM>
    struct Ackley {
      __device__ static dual::DualNumber
      evaluate(const dual::DualNumber* x)
      {
        return ackley<DIM>(x);
      }
      __host__ __device__ static double
      evaluate(const double* x)
      {
        return ackley<DIM>(x);
      }
    };

    // Goldstein-Price Function
    //   f(x,y) = [1+(x+y+1)^2 (19-14x+3x^2-14y+6xy+3y^2)]
    //            [30+(2x-3y)^2 (18-32x+12x^2+48y-36xy+27y^2)]
    template <int DIM>
    __device__ dual::DualNumber
    goldstein_price(const dual::DualNumber* x)
    {
      static_assert(DIM == 2,
                    "Goldstein-Price is defined for 2 dimensions only.");
      dual::DualNumber x1 = x[0];
      dual::DualNumber x2 = x[1];
      dual::DualNumber term1 =
        dual::DualNumber(1.0) +
        dual::pow(x1 + x2 + 1.0, 2) *
          (19.0 - 14.0 * x1 + 3.0 * dual::pow(x1, 2) - 14.0 * x2 +
           6.0 * x1 * x2 + 3.0 * dual::pow(x2, 2));
      dual::DualNumber term2 =
        dual::DualNumber(30.0) +
        dual::pow(2.0 * x1 - 3.0 * x2, 2) *
          (18.0 - 32.0 * x1 + 12.0 * dual::pow(x1, 2) + 48.0 * x2 -
           36.0 * x1 * x2 + 27.0 * dual::pow(x2, 2));
      return term1 * term2;
    }

    template <int DIM>
    __host__ __device__ double
    goldstein_price(const double* x)
    {
      static_assert(DIM == 2,
                    "Goldstein-Price is defined for 2 dimensions only.");
      double x1 = x[0];
      double x2 = x[1];
      double term1 = 1.0 + pow(x1 + x2 + 1.0, 2) *
                             (19.0 - 14.0 * x1 + 3.0 * pow(x1, 2) - 14.0 * x2 +
                              6.0 * x1 * x2 + 3.0 * pow(x2, 2));
      double term2 = 30.0 + pow(2.0 * x1 - 3.0 * x2, 2) *
                              (18.0 - 32.0 * x1 + 12.0 * pow(x1, 2) +
                               48.0 * x2 - 36.0 * x1 * x2 + 27.0 * pow(x2, 2));
      return term1 * term2;
    }

    template <int DIM>
    struct GoldsteinPrice {
      __device__ static dual::DualNumber
      evaluate(const dual::DualNumber* x)
      {
        return goldstein_price<DIM>(x);
      }
      __host__ __device__ static double
      evaluate(const double* x)
      {
        return goldstein_price<DIM>(x);
      }
    };

    // Eggholder Function
    //   f(x,y) = -(y+47) sin\Bigl(\sqrt{\Bigl|x/2+y+47\Bigr|}\Bigr)
    //            - x sin\Bigl(\sqrt{\Bigl|x-(y+47)\Bigr|}\Bigr)

    template <int DIM>
    __device__ dual::DualNumber
    eggholder(const dual::DualNumber* x)
    {
      static_assert(DIM == 2, "Eggholder is defined for 2 dimensions only.");
      dual::DualNumber x1 = x[0], x2 = x[1];
      // Use (0 - value) in place of unary minus
      dual::DualNumber term1 =
        (dual::DualNumber(0.0) - (x2 + dual::DualNumber(47.0))) *
        dual::sin(dual::sqrt(
          dual_abs(x1 / dual::DualNumber(2.0) + x2 + dual::DualNumber(47.0))));
      dual::DualNumber term2 =
        (dual::DualNumber(0.0) - x1) *
        dual::sin(dual::sqrt(dual_abs(x1 - (x2 + dual::DualNumber(47.0)))));
      return term1 + term2;
    }

    template <int DIM>
    __device__ double
    eggholder(const double* x)
    {
      static_assert(DIM == 2, "Eggholder is defined for 2 dimensions only.");
      double x1 = x[0];
      double x2 = x[1];
      double term1 = -(x2 + 47.0) * sin(sqrt(fabs(x1 / 2.0 + x2 + 47.0)));
      double term2 = -x1 * sin(sqrt(fabs(x1 - (x2 + 47.0))));
      return term1 + term2;
    }

    template <int DIM>
    struct Eggholder {
      __host__ __device__ static dual::DualNumber
      evaluate(const dual::DualNumber* x)
      {
        return eggholder<DIM>(x);
      }
      __host__ __device__ static double
      evaluate(const double* x)
      {
        return eggholder<DIM>(x);
      }
    };

    // Himmelblau's Function (2D only)
    template <int DIM>
    __device__ dual::DualNumber
    himmelblau(const dual::DualNumber* x)
    {
      static_assert(DIM == 2,
                    "Himmelblau's function is defined for 2 dimensions only.");
      dual::DualNumber x1 = x[0], x2 = x[1];
      dual::DualNumber term1 =
        dual::pow(x1 * x1 + x2 - dual::DualNumber(11.0), 2);
      dual::DualNumber term2 =
        dual::pow(x1 + x2 * x2 - dual::DualNumber(7.0), 2);
      return term1 + term2;
    }

    template <int DIM>
    __host__ __device__ double
    himmelblau(const double* x)
    {
      static_assert(DIM == 2,
                    "Himmelblau's function is defined for 2 dimensions only.");
      double x1 = x[0], x2 = x[1];
      double term1 = pow(x1 * x1 + x2 - 11.0, 2);
      double term2 = pow(x1 + x2 * x2 - 7.0, 2);
      return term1 + term2;
    }

    template <int DIM>
    struct Himmelblau {
      __host__ __device__ static dual::DualNumber
      evaluate(const dual::DualNumber* x)
      {
        return himmelblau<DIM>(x);
      }
      __host__ __device__ static double
      evaluate(const double* x)
      {
        return himmelblau<DIM>(x);
      }
    };

    template <typename Function, int DIM>
    __device__ void
    calculateGradientUsingAD(double* x, double* gradient)
    {
      dual::DualNumber xDual[DIM];

      for (int i = 0; i < DIM;
           ++i) { // // iterate through each dimension (vairbale)
        xDual[i] = dual::DualNumber(x[i], 0.0);
      }

      // calculate the partial derivative of  each dimension
      for (int i = 0; i < DIM; ++i) {
        xDual[i].dual = 1.0; // derivative w.r.t. dimension i
        dual::DualNumber result =
          Function::evaluate(xDual); // evaluate the function using AD
        gradient[i] = result.dual;   // store derivative
        // printf("\nxDual[%d]: %f, grad[%d]: %f
        // ",i,xDual[i].real,i,gradient[i]);
        xDual[i].dual = 0.0;
      }
    }

    __device__ double
    generate_random_double(unsigned int seed, double lower, double upper)
    {
      hiprandState state;
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      hiprand_init(
        seed, idx, 0, &state); // initialize cuRAND with unique sequence number
      return lower + (upper + (-lower)) * hiprand_uniform_double(&state);
      // return -5.0 + (-3.0 + 5.12) * hiprand_uniform_double(&state); // return
      // scaled double
    }

    template <typename Function, int DIM>
    __device__ double
    line_search(double f0, const double* x, const double* p, const double* g)
    {
      const double c1 = 0.3;
      double alpha = 1.0;
      double ddir = dot_product_device(g, p, DIM);
      double xTemp[DIM];
      for (int i = 0; i < 20; i++) {
        for (int j = 0; j < DIM; j++) {
          xTemp[j] = x[j] + alpha * p[j];
        }
        double f1 = Function::evaluate(xTemp);
        if (f1 <= f0 + c1 * alpha * ddir)
          break;
        alpha *= 0.5;
      }
      return alpha;
    }

} // util namespace end

__device__ __forceinline__ double
atomicMinDouble(double* addr, double val)
{
  // reinterpret the address as 64‑bit unsigned
  unsigned long long* ptr = reinterpret_cast<unsigned long long*>(addr);
  unsigned long long old_bits = *ptr, assumed_bits;

  do {
    assumed_bits = old_bits;
    double old_val = __longlong_as_double(assumed_bits);
    // if the current value is already <= our candidate, nothing to do
    if (old_val <= val)
      break;
    // else try to swap in the new min value’s bit‐pattern
    unsigned long long new_bits = __double_as_longlong(val);
    old_bits = atomicCAS(ptr, assumed_bits, new_bits);
  } while (assumed_bits != old_bits);

  // return the previous minimum
  return __longlong_as_double(old_bits);
}

// kernel #1: initialize X, V, pBest; atomically seed gBestVal/gBestX
template <typename Function, int DIM>
__global__ void
psoInitKernel(Function func,
              double lower,
              double upper,
              double* X,
              double* V,
              double* pBestX,
              double* pBestVal,
              double* gBestX,
              double* gBestVal,
              int N,
              uint64_t seed)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N)
    return;
  const double vel_range = (upper - lower) * 0.1;
  // const unsigned int seed = 1234u;
  // uint64_t counter = seed ^ (uint64_t)tid;
  unsigned int basePos = 1234u ^ (i * 0x9e3779b9);
  unsigned int baseVel = 2468u ^ (i * 0x7f4a7c15);
  // init position & velocity
  for (int d = 0; d < DIM; ++d) {
    unsigned int seedX = basePos ^ (d * 0x85ebca6bu);
    unsigned int seedV = baseVel ^ (d * 0xc2b2ae35u);
    double rx = util::generate_random_double(seedX, lower, upper);
    double rv = util::generate_random_double(seedV, -vel_range, vel_range);

    X[i * DIM + d] = rx;
    V[i * DIM + d] = rv;
    pBestX[i * DIM + d] = rx;
  }

  // eval personal best
  double fval = Function::evaluate(&X[i * DIM]);
  pBestVal[i] = fval;

  // atomic update of global best
  double oldGB = atomicMinDouble(gBestVal, fval);
  if (fval < oldGB) {
    // we’re the new champion: copy pBestX into gBestX
    for (int d = 0; d < DIM; ++d)
      gBestX[d] = pBestX[i * DIM + d];
  }
}

// kernel #2: one PSO iteration (velocity+position update, personal & global
// best)
template <typename Function, int DIM>
__global__ void
psoIterKernel(Function func,
              double lower,
              double upper,
              double w,  // weight inertia
              double c1, // cognitive coefficient
              double c2, // social coefficient
              double* X,
              double* V,
              double* pBestX,
              double* pBestVal,
              double* gBestX,
              double* gBestVal,
              double* traj, // pass nullptr if not saving
              bool saveTraj,
              int N,
              int iter,
              uint64_t seed) // iteration index, for RNG diversification
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N)
    return;
  // const unsigned int seedBase = 5678u + iter*2;

  unsigned int base = 5678u + iter * 1315423911u;
  // update velocity & position
  for (int d = 0; d < DIM; ++d) {
    unsigned int seed1 = base ^ (i * d + 0);
    unsigned int seed2 = base ^ (i * d + 1);

    double r1 = util::generate_random_double(seed1, 0.0, 1.0);
    double r2 = util::generate_random_double(seed2, 0.0, 1.0);

    double xi = X[i * DIM + d];
    double vi = V[i * DIM + d];
    double pb = pBestX[i * DIM + d];
    double gb = gBestX[d];

    double nv = w * vi +
                c1 * r1 * (pb - xi)    // “cognitive” pull toward personal best
                + c2 * r2 * (gb - xi); // “social” pull toward global best
    double nx = xi + nv;

    V[i * DIM + d] = nv;
    X[i * DIM + d] = nx;

    if (saveTraj) {
      // traj is laid out [iter][i][d]
      size_t idx = size_t(iter) * N * DIM + i * DIM + d;
      traj[idx] = nx;
    }
  }

  // evaluate at new position
  double fval = Function::evaluate(&X[i * DIM]);

  // personal best? no atomic needed, it's a private best position
  if (fval < pBestVal[i]) {
    pBestVal[i] = fval;
    for (int d = 0; d < DIM; ++d)
      pBestX[i * DIM + d] = X[i * DIM + d];
  }

  // global best?
  double oldGB = atomicMinDouble(gBestVal, fval);
  if (fval < oldGB) {
    for (int d = 0; d < DIM; ++d)
      gBestX[d] = X[i * DIM + d];
  }
  /*printf("it %d gBestVal = %.6f  at gBestX = [",i,fval);
  for (int d = 0; d < DIM; ++d)
      printf(" %8.4f", gBestX[d]);
  printf(" ]\n");*/
}

// const int MAX_ITER = 64;

__device__ int d_stopFlag;       // 0 = keep going; 1 = stop immediately
__device__ int d_convergedCount; // how many threads have converged?
__device__ int d_threadsRemaining;

template <typename Function, int DIM, unsigned int blockSize>
__global__ void
optimizeKernel(
  double lower,
  double upper,
  const double* __restrict__ pso_array, // pso initialized positions
  double* deviceResults,
  int* deviceIndices,
  double* deviceCoordinates,
  double* deviceTrajectory,
  int N,
  int MAX_ITER,
  int requiredConverged,
  double tolerance,
  bool save_trajectories = false)
{
  // template<typename Function, int DIM, unsigned int blockSize>
  //__global__ void optimizeKernel(double* devicePoints,double* deviceResults,
  // int N) {
  //__global__ void optimizeKernel(double lower, double upper, double*
  // deviceResults, int* deviceIndices, double* deviceCoordinates, int N, int
  // MAX_ITER) {
  extern __device__ int d_stopFlag;
  extern __device__ int d_threadsRemaining;
  extern __device__ int d_convergedCount;

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= N)
    return;

  // int early_stopping = 0;
  double H[DIM * DIM];
  double g[DIM], x[DIM], x_new[DIM], p[DIM], g_new[DIM], delta_x[DIM],
    delta_g[DIM]; //, new_direction[DIM];
  // double tolerance = 1e-5;
  //  Line Search params

  util::initialize_identity_matrix(H, DIM);
  // unsigned int seed = 135;
  // unsigned int seed = 246;
  // unsigned int seed = 37;

  int num_steps = 0;

  // unsigned int seed = 1;
  // unsigned int seed = 456;
  //  initialize x either from PSO array or (fallback) by RNG
  if (pso_array) {
#pragma unroll
    for (int d = 0; d < DIM; ++d) {
      x[d] = pso_array[idx * DIM + d];
    }
  } else {
    unsigned int seed = 456;
#pragma unroll
    for (int d = 0; d < DIM; ++d) {
      // x[d] = util::statelessUniform(idx,d,1,lower, upper, seed);
      x[d] = util::generate_random_double(seed + idx * DIM + d, lower, upper);
    }
  }
  /*for (int i = 0; i < DIM; ++i) {
      x[i] = util::generate_random_double(seed+idx*DIM+i, lower, upper);//
  devicePoints[i * dim + idx];
      //deviceStartingPositions[idx * DIM + i] = x[i];
      //printf("x[%d]: %f",i,x[i]);//<<std::endl;
  }*/
  double f0 = Function::evaluate(x); // rosenbrock_device(x, DIM);
  deviceResults[idx] = f0;
  double bestVal = f0;
  if (idx == 0) {
    printf("\n\nf0 = %f", f0);
  }
  util::calculateGradientUsingAD<Function, DIM>(x, g);
  for (int iter = 0; iter < MAX_ITER; ++iter) {
    // check if somebody already asked to stop
    if (atomicAdd(&d_stopFlag, 0) !=
        0) { // atomicAdd here just to get a strong read-barrier
      // CUDA will fetch a coherent copy of the integer from global memory.
      // as soon as one thread writes 1 into d_stopFlag via atomicExch,
      // the next time any thread does atomicAdd(&d_stopFlag, 0) it’ll see 1 and
      // break.
      // printf("thread %d get outta dodge cuz we converged...", idx);
      break;
    }
    num_steps++;
    // printf("idx = %d", idx);
    // printf("x[0] = %f", x[0]);
    // printf("\n\n\nit#%d",iter);
    // Function::gradient(x, g, DIM);
    // rosenbrock_gradient_device(x, g, DIM);
    // util::calculateGradientUsingAD<Function, DIM>(x, g);

    // d0 = 0.0;
    //  compute the search direction p = -H * g
    for (int i = 0; i < DIM; i++) {
      double sum = 0.0;
      for (int j = 0; j < DIM; j++) {
        sum += H[i * DIM + j] * g[j]; // i * dim + j since H is flattened arr[]
      }
      p[i] = -sum;
    }

    // use the alpha obtained from the line search
    double alpha = util::line_search<Function, DIM>(bestVal, x, p, g);
    if (alpha == 0.0) {
      printf("Alpha is zero, no movement in iteration=%d\n", iter);
      alpha = 1e-3;
    }
    // printf("alpha before updat at it%de: %f", iter, alpha);

    // update current point x by taking a step size of alpha in the direction p
    for (int i = 0; i < DIM; ++i) {
      x_new[i] = x[i] + alpha * p[i];
      delta_x[i] = x_new[i] - x[i];
      // printf("\nnewx[%d]: %f",i,x_new[i]);
    }

    double fnew = Function::evaluate(x_new);
    // get the new gradient g_new at x_new
    util::calculateGradientUsingAD<Function, DIM>(x_new, g_new);

    // calculate new delta_x and delta_g
    for (int i = 0; i < DIM; ++i) {
      delta_g[i] =
        g_new[i] - g[i]; // difference in gradient at the new point vs old point
    }

    // calculate the the dot product between the change in x and change in
    // gradient using new point
    double delta_dot = util::dot_product_device(delta_x, delta_g, DIM);

    // bfgs update on H
    util::bfgs_update<DIM>(H, delta_x, delta_g, delta_dot);
    // only update x and g for next iteration if the new minima is smaller than
    // previous
    // double min = Function::evaluate(x_new);//rosenbrock_device(x_new, DIM);
    if (fnew < bestVal) {
      bestVal = fnew;
      for (int i = 0; i < DIM; ++i) {
        x[i] = x_new[i];
        g[i] = g_new[i];
      }
    }

    double grad_norm = 0.0;
    for (int i = 0; i < DIM; ++i) {
      grad_norm += g[i] * g[i];
    }
    grad_norm = sqrt(grad_norm);
    if (grad_norm < tolerance) {
      // atomically increment the “how many have converged” counter:
      int oldCount = atomicAdd(&d_convergedCount, 1);
      int newCount = oldCount + 1;
      double fcurr = Function::evaluate(x);
      // now newCount == (number of threads that have now reported convergence).
      // printf("\nconverged for %d at iter=%d); f = %.6f;",idx, iter,fcurr);
      // for (int d = 0; d < DIM; ++d) { printf(" % .6f", x[d]);}
      // printf(" ]\n");

      // if we just hit the threshold K set by the user, the VERY FIRST thread
      // to do so sets d_stopFlag=1 so everyone else exits on their next check.
      if (newCount == requiredConverged) {
        // flip the global stop flag
        atomicExch(&d_stopFlag, 1);
        __threadfence();
        printf(
          "\nThread %d is the %d%s converged thread (iter=%d); fn = %.6f.\n",
          idx,
          newCount,
          (newCount == 1 ? "st" :
           newCount == 2 ? "nd" :
           newCount == 3 ? "rd" :
                           "th"),
          iter,
          fcurr);
      }
      // in _any_ case (whether we were the kth or not),
      // we are individually “done,” so break
      break;
    }

    //  deviceTrajectory layout: idx * (MAX_ITER * DIM) + iter * DIM + i
    if (save_trajectories) {
      for (int i = 0; i < DIM; i++) {
        deviceTrajectory[idx * (MAX_ITER * DIM) + iter * DIM + i] = x[i];
      }
    }

    // for(int i=0; i<DIM; ++i) {x[i] = x_new[i];}
  } // end outer for

  if (atomicAdd(&d_stopFlag, 0) == 0) {
    // We reached MAX_ITER without hitting grad_norm < tol and without ever
    // seeing the flag. That means we “fully looped” and never decremented
    // above. Subtract now:
    int oldCount2 = atomicSub(&d_threadsRemaining, 1);
  }
  bestVal = Function::evaluate(x); // rosenbrock_device(x, DIM);
  // printf("\nmax iterations reached, predicted minima = %f\n", minima);
  deviceResults[idx] = bestVal;
  deviceIndices[idx] = idx;
  for (int i = 0; i < DIM; ++i) {
    deviceCoordinates[idx * DIM + i] = x[i];
  }
} // end optimizerKernel

bool
askUser2saveTrajectories()
{
  std::cout << "Save optimization trajectories? (y/n): ";
  char ans;
  std::cin >> ans;
  std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
  return (ans == 'y' || ans == 'Y');
}

void
createOutputDirs(const std::string& path)
{
  std::filesystem::create_directories(path);
}

hipError_t
writeTrajectoryData(double* hostTrajectory,
                    int N,
                    int MAX_ITER,
                    int DIM,
                    const std::string& fun_name,
                    const std::string& basePath)
{
  // construct the directory path and create it.
  std::string dirPath = basePath + "/" + fun_name + "/" + std::to_string(DIM) +
                        "d/" + std::to_string(MAX_ITER * N) + "/trajectories";
  std::filesystem::create_directories(dirPath);
  // createOutputDirs(dirPath);

  // the final filename.
  std::string filename = dirPath + "/" + std::to_string(MAX_ITER) + "it_" +
                         std::to_string(N) + ".tsv";

  std::ofstream stepOut(filename);
  stepOut << "OptIndex\tStep";
  for (int d = 0; d < DIM; d++)
    stepOut << "\tX_" << d;
  stepOut << "\n";
  stepOut << std::scientific << std::setprecision(17);
  for (int i = 0; i < N; i++) {
    for (int it = 0; it < MAX_ITER; it++) {
      stepOut << i << "\t" << it;
      for (int d = 0; d < DIM; d++) {
        stepOut << "\t" << hostTrajectory[i * (MAX_ITER * DIM) + it * DIM + d];
      }
      stepOut << "\n";
    }
  }
  stepOut.close();
  return hipSuccess;
}

template <typename Function, int DIM>
hipError_t
launchOptimizeKernel(double lower,
                     double upper,
                     double* hostResults,
                     int* hostIndices,
                     double* hostCoordinates,
                     int N,
                     int MAX_ITER,
                     int PSO_ITER,
                     int requiredConverged,
                     std::string fun_name,
                     double tolerance)
{
  int blockSize, minGridSize;
  hipOccupancyMaxPotentialBlockSize(
    &minGridSize, &blockSize, optimizeKernel<Function, DIM, 128>, 0, N);
  printf("Recommended block size: %d\n", blockSize);

  bool save_trajectories = askUser2saveTrajectories();
  double* deviceTrajectory = nullptr;
  double *dX = nullptr, *dV = nullptr, *dPBestVal = nullptr, *dGBestX = nullptr,
         *dGBestVal = nullptr, *dPBestX = nullptr;
  float ms_init, ms_pso, ms_opt;
  if (PSO_ITER > 0) {
    // allocate PSO buffers on device
    double *dX, *dV, *dPBestVal, *dGBestX, *dGBestVal;
    hipMalloc(&dX, N * DIM * sizeof(double));
    hipMalloc(&dV, N * DIM * sizeof(double));
    hipMalloc(&dPBestX, N * DIM * sizeof(double));
    hipMalloc(&dPBestVal, N * sizeof(double));
    hipMalloc(&dGBestX, DIM * sizeof(double));
    hipMalloc(&dGBestVal, sizeof(double));
    int zero = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(d_stopFlag), &zero, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_threadsRemaining), &N, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_convergedCount), &zero, sizeof(int));
    // set seed to infinity
    {
      double inf = std::numeric_limits<double>::infinity();
      hipMemcpy(dGBestVal, &inf, sizeof(inf), hipMemcpyHostToDevice);
    }

    dim3 psoBlock(256);
    dim3 psoGrid((N + psoBlock.x - 1) / psoBlock.x);

    // host-side buffers for printing
    double hostGBestVal;
    std::vector<double> hostGBestX(DIM);

    // PSO‐init Kernel
    hipEvent_t t0, t1;
    hipEventCreate(&t0);
    hipEventCreate(&t1);

    hipEventRecord(t0);
    psoInitKernel<Function, DIM><<<psoGrid, psoBlock>>>(Function(),
                                                        lower,
                                                        upper,
                                                        dX,
                                                        dV,
                                                        dPBestX,
                                                        dPBestVal,
                                                        dGBestX,
                                                        dGBestVal,
                                                        N,
                                                        1234567);
    hipDeviceSynchronize();
    hipEventRecord(t1);
    hipEventSynchronize(t1);

    hipEventElapsedTime(&ms_init, t0, t1);
    printf("PSO‑Init Kernel execution time = %.4f ms\n", ms_init);

    // copy back and print initial global best
    hipMemcpy(
      &hostGBestVal, dGBestVal, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(
      hostGBestX.data(), dGBestX, DIM * sizeof(double), hipMemcpyDeviceToHost);

    printf("Initial PSO gBestVal = %.6e at gBestX = [", hostGBestVal);
    for (int d = 0; d < DIM; ++d)
      printf(" %.4f", hostGBestX[d]);
    printf(" ]\n\n");

    // PSO iterations (each timed + host‐print) ––
    const double w = 0.5, c1 = 1.2, c2 = 1.5;
    for (int iter = 1; iter < PSO_ITER + 1; ++iter) {
      hipEventRecord(t0);
      psoIterKernel<Function, DIM><<<psoGrid, psoBlock>>>(Function(),
                                                          lower,
                                                          upper,
                                                          w,
                                                          c1,
                                                          c2,
                                                          dX,
                                                          dV,
                                                          dPBestX,
                                                          dPBestVal,
                                                          dGBestX,
                                                          dGBestVal,
                                                          /*traj=*/nullptr,
                                                          /*saveTraj=*/false,
                                                          N,
                                                          iter,
                                                          1234567);
      hipDeviceSynchronize();
      hipEventRecord(t1);
      hipEventSynchronize(t1);

      float ms_iter = 0;
      hipEventElapsedTime(&ms_iter, t0, t1);
      hipMemcpy(
        &hostGBestVal, dGBestVal, sizeof(double), hipMemcpyDeviceToHost);
      hipMemcpy(hostGBestX.data(),
                 dGBestX,
                 DIM * sizeof(double),
                 hipMemcpyDeviceToHost);

      printf("PSO‑Iter %2d execution time = %.3f ms   gBestVal = %.6e at [",
             iter,
             ms_iter,
             hostGBestVal);
      for (int d = 0; d < DIM; ++d)
        printf(" %.4f", hostGBestX[d]);
      printf(" ]\n");
      ms_pso += ms_iter;
    } // end pso loop
    printf("total pso time = %.3f\n", ms_pso + ms_init);

    hipEventDestroy(t0);
    hipEventDestroy(t1);
  } // end if pso_iter > 0

  // prepare optimizer buffers & copy hostResults→device ––
  double* deviceResults;
  int* deviceIndices;
  double* deviceCoords;
  hipcub::KeyValuePair<int, double>* deviceArgMin;
  hipMalloc(&deviceResults, N * sizeof(double));
  hipMalloc(&deviceIndices, N * sizeof(int));
  hipMalloc(&deviceCoords, N * DIM * sizeof(double));
  hipMalloc(&deviceArgMin, sizeof(*deviceArgMin));
  hipMemcpy(
    deviceResults, hostResults, N * sizeof(double), hipMemcpyHostToDevice);

  dim3 optBlock(blockSize);
  dim3 optGrid((N + blockSize - 1) / blockSize);

  // optimizeKernel time
  hipEvent_t startOpt, stopOpt;
  hipEventCreate(&startOpt);
  hipEventCreate(&stopOpt);
  hipEventRecord(startOpt);

  if (save_trajectories) {
    hipMalloc(&deviceTrajectory, N * MAX_ITER * DIM * sizeof(double));
    optimizeKernel<Function, DIM, 128>
      <<<optGrid, optBlock>>>(lower,
                              upper,
                              dPBestX,
                              deviceResults,
                              deviceIndices,
                              deviceCoords,
                              deviceTrajectory,
                              N,
                              MAX_ITER,
                              requiredConverged,
                              tolerance,
                              /*saveTraj=*/true);
  } else {
    optimizeKernel<Function, DIM, 128><<<optGrid, optBlock>>>(lower,
                                                              upper,
                                                              dPBestX,
                                                              deviceResults,
                                                              deviceIndices,
                                                              deviceCoords,
                                                              /*traj=*/nullptr,
                                                              N,
                                                              MAX_ITER,
                                                              requiredConverged,
                                                              tolerance);
  }
  hipDeviceSynchronize();
  hipEventRecord(stopOpt);
  hipEventSynchronize(stopOpt);

  hipEventElapsedTime(&ms_opt, startOpt, stopOpt);
  printf("\nOptimization Kernel execution time = %.3f ms\n", ms_opt);

  hipEventDestroy(startOpt);
  hipEventDestroy(stopOpt);

  // ArgMin & final print
  void* d_temp_storage = nullptr;
  size_t temp_bytes = 0;
  hipcub::DeviceReduce::ArgMin(
    d_temp_storage, temp_bytes, deviceResults, deviceArgMin, N);
  hipMalloc(&d_temp_storage, temp_bytes);
  hipcub::DeviceReduce::ArgMin(
    d_temp_storage, temp_bytes, deviceResults, deviceArgMin, N);

  hipcub::KeyValuePair<int, double> h_argMin;
  hipMemcpy(&h_argMin, deviceArgMin, sizeof(h_argMin), hipMemcpyDeviceToHost);

  int globalMinIndex = h_argMin.key;
  double globalMin = h_argMin.value;

  printf("\nFinal Global Minima: %f  (index %d)\n", globalMin, globalMinIndex);

  hipMemcpy(hostCoordinates,
             deviceCoords + size_t(globalMinIndex) * DIM,
             DIM * sizeof(double),
             hipMemcpyDeviceToHost);

  printf("Coordinates: [");
  for (int d = 0; d < DIM; ++d)
    printf(" %.7f", hostCoordinates[d]);
  printf(" ]\n");
  double error;
  double fStar = 0.0; // f(x*) for high dim functions
  double fVal = globalMin;

  if (std::abs(fStar) > 0.0)
    error = std::abs(fVal - fStar) / std::abs(fStar);
  else
    error = std::abs(fVal);
  // Append results to a .tsv file in scientific notation
  {
    std::string filename = std::to_string(DIM) + "d_results.tsv";
    std::ofstream outfile(filename, std::ios::app);
    double time_seconds = std::numeric_limits<double>::infinity();
    if (PSO_ITER > 0) {
      time_seconds = (ms_init + ms_pso + ms_opt);
      printf("total time = pso + bfgs = total time = %0.4f ms\n", time_seconds);
    } else {
      time_seconds = ms_opt;
      printf("bfgs time = total time = %.4f ms\n", time_seconds);
    }
    outfile << fun_name << "\t" << N << "\t" << MAX_ITER << "\t" << PSO_ITER
            << "\t" << time_seconds << "\t" << error << "\t" << std::scientific
            << globalMin << "\t";
    for (int i = 0; i < DIM; i++) {
      outfile << hostCoordinates[i];
      if (i < DIM - 1)
        outfile << "\t";
    }
    outfile << "\n";
    outfile.close();
    printf("results are saved to %s", filename.c_str());
  }
  if (PSO_ITER > 0) {
    hipFree(dX);
    hipFree(dV);
    hipFree(dPBestX);
    hipFree(dPBestVal);
    hipFree(dGBestX);
    hipFree(dGBestVal);
  }

  hipFree(deviceResults);
  hipFree(deviceIndices);
  hipFree(deviceCoords);
  hipFree(deviceArgMin);
  hipFree(d_temp_storage);

  return hipSuccess;
} // end launcher

template <typename Function, int DIM>
void
runOptimizationKernel(double lower,
                      double upper,
                      double* hostResults,
                      int* hostIndices,
                      double* hostCoordinates,
                      int N,
                      int MAX_ITER,
                      int PSO_ITERS,
                      int requiredConverged,
                      std::string fun_name,
                      double tolerance)
{
  // void runOptimizationKernel(double* hostResults, int N, int dim) {
  /*printf("first 20 hostResults\n");
  for(int i=0;i<20;i++) {
     printf(" %f ",hostResults[i]);
  }
  printf("\n");
  */
  hipError_t error = launchOptimizeKernel<Function, DIM>(lower,
                                                          upper,
                                                          hostResults,
                                                          hostIndices,
                                                          hostCoordinates,
                                                          N,
                                                          MAX_ITER,
                                                          PSO_ITERS,
                                                          requiredConverged,
                                                          fun_name,
                                                          tolerance);
  if (error != hipSuccess) {
    printf("CUDA error: %s", hipGetErrorString(error));
  } else {
    printf("\nSuccess!! No Error!\n");
  }
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // printf("Sorting the array with %d elements... ", N);
  hipEventRecord(start);
  // quickSort(hostResults, 0, N - 1);
  hipEventRecord(stop);
  float milli = 0;
  hipEventElapsedTime(&milli, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  // printf("took %f ms\n",  milli);

  /*printf("first 20 function values in hostResults\n");
  for(int i=0;i<20;i++) {
     printf(" %f ",hostResults[i]);
  }*/
  printf("\n");
  // hipMemGetInfo
}

template <int dim>
void
selectAndRunOptimization(double lower,
                         double upper,
                         double* hostResults,
                         int* hostIndices,
                         double* hostCoordinates,
                         int N,
                         int MAX_ITER,
                         int PSO_ITERS,
                         int requiredConverged,
                         double tolerance)
{
  int choice;
  std::cout << "\nSelect function to optimize:\n"
            << " 1. Rosenbrock\n"
            << " 2. Rastrigin\n"
            << " 3. Ackley\n";
  // Only show 2D-only options when dim == 2.
  if constexpr (dim == 2) {
    std::cout << " 4. GoldsteinPrice\n"
              << " 5. Eggholder\n"
              << " 6. Himmelblau\n";
  }
  std::cout
    << " 7. Custom (user-defined objective via expression or kernel file)\n"
    << "Choice: ";
  std::cin >> choice;
  std::cin.ignore();

  switch (choice) {
    case 1:
      std::cout << "\n\n\tRosenbrock Function\n" << std::endl;
      runOptimizationKernel<util::Rosenbrock<dim>, dim>(lower,
                                                        upper,
                                                        hostResults,
                                                        hostIndices,
                                                        hostCoordinates,
                                                        N,
                                                        MAX_ITER,
                                                        PSO_ITERS,
                                                        requiredConverged,
                                                        "rosenbrock",
                                                        tolerance);
      break;
    case 2:
      std::cout << "\n\n\tRastrigin Function\n" << std::endl;
      runOptimizationKernel<util::Rastrigin<dim>, dim>(lower,
                                                       upper,
                                                       hostResults,
                                                       hostIndices,
                                                       hostCoordinates,
                                                       N,
                                                       MAX_ITER,
                                                       PSO_ITERS,
                                                       requiredConverged,
                                                       "rastrigin",
                                                       tolerance);
      break;
    case 3:
      std::cout << "\n\n\tAckley Function\n" << std::endl;
      runOptimizationKernel<util::Ackley<dim>, dim>(lower,
                                                    upper,
                                                    hostResults,
                                                    hostIndices,
                                                    hostCoordinates,
                                                    N,
                                                    MAX_ITER,
                                                    PSO_ITERS,
                                                    requiredConverged,
                                                    "ackley",
                                                    tolerance);
      break;
    case 4:
      if constexpr (dim != 2) {
        std::cerr
          << "Error: GoldsteinPrice is defined for 2 dimensions only.\n";
      } else {
        std::cout << "\n\n\tGoldsteinPrice Function\n" << std::endl;
        runOptimizationKernel<util::GoldsteinPrice<dim>, dim>(lower,
                                                              upper,
                                                              hostResults,
                                                              hostIndices,
                                                              hostCoordinates,
                                                              N,
                                                              MAX_ITER,
                                                              PSO_ITERS,
                                                              requiredConverged,
                                                              "goldstein",
                                                              tolerance);
      }
      break;
    case 5:
      if constexpr (dim != 2) {
        std::cerr << "Error: Eggholder is defined for 2 dimensions only.\n";
      } else {
        std::cout << "\n\n\tEggholder Function\n" << std::endl;
        runOptimizationKernel<util::Eggholder<dim>, dim>(lower,
                                                         upper,
                                                         hostResults,
                                                         hostIndices,
                                                         hostCoordinates,
                                                         N,
                                                         MAX_ITER,
                                                         PSO_ITERS,
                                                         requiredConverged,
                                                         "eggholder",
                                                         tolerance);
      }
      break;
    case 6:
      if constexpr (dim != 2) {
        std::cerr << "Error: Himmelblau is defined for 2 dimensions only.\n";
      } else {
        std::cout << "\n\n\tHimmelblau Function\n" << std::endl;
        runOptimizationKernel<util::Himmelblau<dim>, dim>(lower,
                                                          upper,
                                                          hostResults,
                                                          hostIndices,
                                                          hostCoordinates,
                                                          N,
                                                          MAX_ITER,
                                                          PSO_ITERS,
                                                          requiredConverged,
                                                          "himmelblau",
                                                          tolerance);
      }
      break;
    case 7:
      std::cout << "\n\n\tCustom User-Defined Function\n" << std::endl;
      // for a more complex custom function, one option is to let the user
      // provide a path to a cuda file and compile it at runtime.
      // runOptimizationKernel<UserDefined<dim>, dim>(lower, upper, hostResults,
      // hostIndices,
      //                                             hostCoordinates, N,
      //                                             MAX_ITER);
      break;
    default:
      std::cerr << "Invalid selection!\n";
      exit(1);
  }
}

#ifndef UNIT_TEST
int
main(int argc, char* argv[])
{
  printf("Production main() running\n");
  if (argc != 8) {
    std::cerr << "Usage: " << argv[0]
              << " <lower_bound> <upper_bound> <max_iter> <pso_iters> "
                 "<converged> <number_of_optimizations> <tolerance>\n";
    return 1;
  }
  double lower = std::atof(argv[1]);
  double upper = std::atof(argv[2]);
  int MAX_ITER = std::stoi(argv[3]);
  int PSO_ITERS = std::stoi(argv[4]);
  int requiredConverged = std::stoi(argv[5]);
  int N = std::stoi(argv[6]);
  double tolerance = std::stod(argv[7]);
  std::cout << "Tolerance: " << std::setprecision(10) << tolerance << "\n";

  // const size_t N =
  // 128*4;//1024*128*16;//pow(10,5.5);//128*1024*3;//*1024*128;
  const int dim = 5;
  double hostResults[N]; // = new double[N];
  std::cout << "number of optimizations = " << N << " max_iter = " << MAX_ITER
            << " dim = " << dim << std::endl;

  int hostIndices[N];
  double hostCoordinates[dim];
  double f0 = 333777; // initial function value

  // logic to set the stact size limit to 65 kB per thread
  size_t currentStackSize = 0;
  hipDeviceGetLimit(&currentStackSize, hipLimitStackSize);
  printf("Current stack size: %zu bytes\n", currentStackSize);
  size_t newStackSize = 64 * 1024; // 65 kB
  hipError_t err = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
  if (err != hipSuccess) {
    printf("hipDeviceSetLimit error: %s\n", hipGetErrorString(err));
    return 1;
  } else {
    printf("Successfully set stack size to %zu bytes\n", newStackSize);
  } // end stack size limit

  char cont = 'y';
  while (cont == 'y' || cont == 'Y') {
    for (int i = 0; i < N; i++) {
      hostResults[i] = f0;
    }
    selectAndRunOptimization<dim>(lower,
                                  upper,
                                  hostResults,
                                  hostIndices,
                                  hostCoordinates,
                                  N,
                                  MAX_ITER,
                                  PSO_ITERS,
                                  requiredConverged,
                                  tolerance);
    std::cout << "\nDo you want to optimize another function? (y/n): ";
    std::cin >> cont;
    std::cin.ignore();
  }

  // for(int i=0; i<N; i++) {
  //     hostResults[i] = f0;
  // }
  // selectAndRunOptimization<dim>(lower, upper, hostResults, hostIndices,
  // hostCoordinates, N, MAX_ITER);
  return 0;
}
#endif
