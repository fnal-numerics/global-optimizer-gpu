#include "cuda_buffer.cuh"

// allocate uninitialized buffer
cuda_buffer::cuda_buffer(std::size_t n)
  : d(nullptr), sz(n)
{
  hipError_t status = hipMalloc(&d, sz * sizeof(double));
  if (status != hipSuccess) {
    throw cuda_exception<3>("hipMalloc failed");
  }
}

// destructor
cuda_buffer::~cuda_buffer() {
  if (d) hipFree(d);
}

// copy ctor
cuda_buffer::cuda_buffer(cuda_buffer const& o)
  : d(nullptr), sz(o.sz)
{
  hipError_t st = hipMalloc(&d, sz * sizeof(double));
  if (st != hipSuccess) {
    throw cuda_exception<3>("hipMalloc failed in copy ctor");
  }
  st = hipMemcpy(d, o.d, sz * sizeof(double),
                  hipMemcpyDeviceToDevice);
  if (st != hipSuccess) {
    throw cuda_exception<4>("hipMemcpy D→D failed in copy ctor");
  }
}

// swap helper
void cuda_buffer::swap(cuda_buffer& o) noexcept {
  std::swap(d, o.d);
  std::swap(sz, o.sz);
}

// copy-assign (copy-and-swap)
cuda_buffer& cuda_buffer::operator=(cuda_buffer const& o) {
  if (this != &o) {
    cuda_buffer tmp(o);
    swap(tmp);
  }
  return *this;
}

// vector-return overload (throws on failure)
std::vector<double> cuda_buffer::copy_to_host() const {
  std::vector<double> out;
  int status = copy_to_host(out);
  if (status != 0)
    throw cuda_exception<4>("hipMemcpy D→H failed");
  return out;
}

// raw-pointer overload (returns 0 on success, <0 on error)
int cuda_buffer::copy_to_host(double* out, std::size_t n) const {
  if (n != sz) return -1;
  if (n > 0 && d) {
    auto st = hipMemcpy(out, d, n * sizeof(double),
                         hipMemcpyDeviceToHost);
    if (st != hipSuccess) return -2;
  }
  return 0;
}

// vector-by-ref overload
int cuda_buffer::copy_to_host(std::vector<double>& out) const {
  out.resize(sz);
  return copy_to_host(out.data(), sz);
}

