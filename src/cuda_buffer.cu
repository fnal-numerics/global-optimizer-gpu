#include "cuda_buffer.cuh"

// allocate uninitialized buffer
cuda_buffer::cuda_buffer(std::size_t n)
  : d(nullptr), sz(n)
{
  hipError_t status = hipMalloc(&d, sz * sizeof(double));
  if (status != hipSuccess) {
    throw cuda_exception<3>("hipMalloc failed");
  }
}

// destructor
cuda_buffer::~cuda_buffer() {
  if (d) hipFree(d);
}

// copy ctor
cuda_buffer::cuda_buffer(cuda_buffer const& u)
  : d(nullptr), sz(u.sz)
{
  hipError_t st = hipMalloc(&d, sz * sizeof(double));
  if (st != hipSuccess) {
    throw cuda_exception<3>("hipMalloc failed in copy ctor");
  }
  st = hipMemcpy(d, u.d, sz * sizeof(double),
                  hipMemcpyDeviceToDevice);
  if (st != hipSuccess) {
    throw cuda_exception<4>("hipMemcpy D→D failed in copy ctor");
  }
}

// copy-assign (copy-and-swap)
cuda_buffer& cuda_buffer::operator=(cuda_buffer const& u) {
  if (this != &u) {
    cuda_buffer tmp(u);
    swap(tmp);
  }
  return *this;
}

// move ctor
cuda_buffer::cuda_buffer(cuda_buffer&& u) noexcept
  : d(u.d), sz(u.sz)
{
  u.d = nullptr;
  u.sz = 0;
}

//  move-assign
cuda_buffer& cuda_buffer::operator=(cuda_buffer&& u) noexcept {
  if (this != &u) {
    if (d) hipFree(d);
    d    = u.d;
    sz   = u.sz;
    u.d  = nullptr;
    u.sz = 0;
  }
  return *this;
}


// vector-return overload (throws on failure)
std::vector<double> cuda_buffer::copy_to_host() const {
  std::vector<double> out;
  int status = copy_to_host(out);
  if (status != 0)
    throw cuda_exception<4>("hipMemcpy D→H failed");
  return out;
}

// raw-pointer overload (returns 0 on success, <0 on error)
int cuda_buffer::copy_to_host(double* out, std::size_t n) const {
  if (n != sz) return -1;
  if (n > 0 && d) {
    auto st = hipMemcpy(out, d, n * sizeof(double),
                         hipMemcpyDeviceToHost);
    if (st != hipSuccess) return -2;
  }
  return 0;
}

// vector-by-ref overload
int cuda_buffer::copy_to_host(std::vector<double>& out) const {
  out.resize(sz);
  return copy_to_host(out.data(), sz);
}

