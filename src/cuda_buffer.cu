#include "cuda_buffer.cuh"

// allocate uninitialized buffer
cuda_buffer::cuda_buffer(std::size_t n)
  : d(nullptr), sz(n)
{
  hipError_t status = hipMalloc(&d, sz * sizeof(double));
  if (status != hipSuccess) {
    throw cuda_exception<3>("hipMalloc failed");
  }
}

// destructor
cuda_buffer::~cuda_buffer() {
  if (d) hipFree(d);
}

