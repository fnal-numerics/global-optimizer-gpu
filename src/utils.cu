#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <string>
namespace util {

  bool
  askUser2saveTrajectories()
  {
    std::cout << "Save optimization trajectories? (y/n): ";
    char ans;
    std::cin >> ans;
    std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
    return (ans == 'y' || ans == 'Y');
  }

  void
  createOutputDirs(const std::string& path)
  {
    std::filesystem::create_directories(path);
  }

  void
  set_stack_size()
  {
    // logic to set the stact size limit to 65 kB per thread
    size_t currentStackSize = 0;
    hipDeviceGetLimit(&currentStackSize, hipLimitStackSize);
    // printf("Current stack size: %zu bytes\n", currentStackSize);
    size_t newStackSize = 64 * 1024; // 65 kB
    hipError_t err = hipDeviceSetLimit(hipLimitStackSize, newStackSize);
    if (err != hipSuccess) {
      printf("hipDeviceSetLimit error: %s\n", hipGetErrorString(err));
      // return 1;
    }
  }

  hipError_t
  writeTrajectoryData(double* hostTrajectory,
                      int N,
                      int MAX_ITER,
                      int DIM,
                      const std::string& fun_name,
                      const std::string& basePath)
  {
    // construct the directory path and create it.
    std::string dirPath = basePath + "/" + fun_name + "/" +
                          std::to_string(DIM) + "d/" +
                          std::to_string(MAX_ITER * N) + "/trajectories";
    std::filesystem::create_directories(dirPath);
    // createOutputDirs(dirPath);

    // the final filename.
    std::string filename = dirPath + "/" + std::to_string(MAX_ITER) + "it_" +
                           std::to_string(N) + ".tsv";

    std::ofstream stepOut(filename);
    stepOut << "OptIndex\tStep";
    for (int d = 0; d < DIM; d++)
      stepOut << "\tX_" << d;
    stepOut << "\n";
    stepOut << std::scientific << std::setprecision(17);
    for (int i = 0; i < N; i++) {
      for (int it = 0; it < MAX_ITER; it++) {
        stepOut << i << "\t" << it;
        for (int d = 0; d < DIM; d++) {
          stepOut << "\t"
                  << hostTrajectory[i * (MAX_ITER * DIM) + it * DIM + d];
        }
        stepOut << "\n";
      }
    }
    stepOut.close();
    return hipSuccess;
  }

  double
  calculate_euclidean_error(const std::string fun_name,
                            const double* coordinates,
                            const int dim)
  {
    double sum_sq = 0.0;

    if (fun_name == "rosenbrock") {
      for (int i = 0; i < dim; i++) {
        double diff = coordinates[i] - 1.0;
        sum_sq += diff * diff;
      }
    } else if (fun_name == "goldstein_price") {
      // Goldstein–Price is only defined in 2D (minimum at (0, -1))
      if (dim != 2) {
        fprintf(stderr, "Error: goldstein_price only defined for dim = 2\n");
        return NAN;
      }
      double dx = coordinates[0] - 0.0;
      double dy = coordinates[1] - (-1.0);
      sum_sq = dx * dx + dy * dy;
    } else if (fun_name == "rastrigin" ||
               fun_name ==
                 "ackley") { // both rastrigin and ackley have the same
                             // coordinates for the global minimum
      for (int i = 0; i < dim; ++i) {
        sum_sq += coordinates[i] * coordinates[i];
      }
    }
    return std::sqrt(sum_sq);
  } // end calculate_euclidean_error

  void
  append_results_2_tsv(const int dim,
                       const int N,
                       const std::string fun_name,
                       float ms_init,
                       float ms_pso,
                       float ms_opt,
                       float ms_rand,
                       const int max_iter,
                       const int pso_iter,
                       const double error,
                       const double globalMin,
                       double* hostCoordinates,
                       const int idx,
                       const int status,
                       const double norm,
                       const int run,
                       const int claimed,
                       const int actual,
                       const int surrendered,
                       const int stopped)
  {
    std::string filename = "zeus_" + std::to_string(dim) + "d_results.tsv";
    std::ofstream outfile(filename, std::ios::app);

    bool file_exists = std::filesystem::exists(filename);
    bool file_empty =
      file_exists ? (std::filesystem::file_size(filename) == 0) : true;
    // std::ofstream outfile(filename, std::ios::app);
    if (!outfile.is_open()) {
      std::cerr << "Error opening file: " << filename << std::endl;
      return;
    }
    // if file is new or empty, let us write the header
    if (file_empty) {
      outfile
        << "fun\trun\tN\tclaimed\tactual\tsurrender\tstopped\tidx\tstatus\t"
           "bfgs_iter\tpso_iter\ttime\terror\tfval\tnorm";
      for (int i = 0; i < dim; i++)
        outfile << "\tcoord_" << i;
      outfile << std::endl;
    } // end if file is empty

    double time_seconds = std::numeric_limits<double>::infinity();
    if (pso_iter > 0) {
      time_seconds = (ms_init + ms_pso + ms_opt + ms_rand);
      // printf("total time = pso + bfgs = total time = %0.4f ms\n",
      // time_seconds);
    } else {
      time_seconds = (ms_opt + ms_rand);
      // printf("bfgs time = total time = %.4f ms\n", time_seconds);
    }
    outfile << fun_name << "\t" << run << "\t" << N << "\t" << claimed << "\t"
            << actual << "\t" << surrendered << "\t" << stopped << "\t" << idx
            << "\t" << status << "\t" << max_iter << "\t" << pso_iter << "\t"
            << time_seconds << "\t" << std::scientific << error << "\t"
            << globalMin << "\t" << norm << "\t";
    for (int i = 0; i < dim; i++) {
      outfile << hostCoordinates[i];
      if (i < dim - 1)
        outfile << "\t";
    }
    outfile << "\n";
    outfile.close();
    // printf("results are saved to %s", filename.c_str());
  } // end append_results_2_tsv

  // https://xorshift.di.unimi.it/splitmix64.c
  // Very fast 64-bit mixer — returns a new 64-bit value each time.
  __device__ inline uint64_t
  splitmix64(uint64_t& x)
  {
    uint64_t z = (x += 0x9e3779b97f4a7c15ULL); // 1 add
    z = (z ^ (z >> 30)) *
        0xbf58476d1ce4e5b9ULL; // 1 shift, 1 xor, 1 64x64 multiplier
    z = (z ^ (z >> 27)) *
        0x94d049bb133111ebULL; // 1 shift, 1, xor, 1 64x64 multiplier
    // printf("split");
    return z ^ (z >> 31); // 1 shift, 1 xor
  }

  // return a random double in [minVal, maxVal)
  __device__ inline double
  random_double(uint64_t& state, double minVal, double maxVal)
  {
    // get 64‐bit random int
    uint64_t z = splitmix64(state);
    // map high 53 bits into [0,1)
    double u =
      (z >> 11) *
      (1.0 /
       9007199254740992.0); // discard lower 11 bits, leaving mantissa width of
                            // IEEE double, then normalize integer into [0,1)
    // scale into [minVal, maxVal)
    return minVal + u * (maxVal - minVal);
  }

  __device__ double
  dot_product_device(const double* a, const double* b, int size)
  {
    double sum = 0.0;
    for (int i = 0; i < size; ++i) {
      sum += a[i] * b[i];
    }
    return sum;
  }

  __device__ void
  outer_product_device(const double* v1,
                       const double* v2,
                       double* result,
                       int size)
  {
    for (int i = 0; i < size; ++i) {
      for (int j = 0; j < size; ++j) {
        int idx = i * size + j;
        if (idx < size * size) {
          result[idx] = v1[i] * v2[j];
        } else {
          printf("outer product out of bounds..\ndim=%d i*size+j=%d\n",
                 size,
                 i * size + j);
        }
      }
    }
  }

  extern "C" {
  __device__ void
  vector_add(const double* a, const double* b, double* result, int size)
  {
    for (int i = 0; i < size; ++i) {
      result[i] = a[i] + b[i];
    }
  }

  __device__ void
  vector_scale(const double* a, double scalar, double* result, int dim)
  {
    for (int i = 0; i < dim; ++i) {
      result[i] = a[i] * scalar;
    }
  }

  } // end extern C

  __device__ void
  initialize_identity_matrix(double* H, int dim)
  {
    for (int i = 0; i < dim; ++i) {
      for (int j = 0; j < dim; ++j) {
        H[i * dim + j] = (i == j) ? 1.0 : 0.0;
      }
    }
  }

  __device__ bool
  valid(double x)
  {
    if (isinf(x)) {
      return false;
    } else if (isnan(x)) {
      return false;
    } else {
      return true;
    }
  }

  __device__ double
  pow2(double x)
  {
    return x * x;
  }

  __device__ void
  initialize_identity_matrix_device(double* H, int n)
  {
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n; ++j) {
        H[i * n + j] = (i == j) ? 1.0 : 0.0;
      }
    }
  }

  template <int DIM>
  __device__ void
  matrix_multiply_device(const double* A, const double* B, double* C)
  {
    for (int i = 0; i < DIM; ++i) {
      for (int j = 0; j < DIM; ++j) {
        double sum = 0.0;
        for (int k = 0; k < DIM; ++k) {
          sum += A[i * DIM + k] * B[k * DIM + j];
        }
        C[i * DIM + j] = sum;
      }
    }
  }

  // function to calculate scalar directional direvative d = g * p
  __device__ double
  directional_derivative(const double* grad, const double* p, int dim)
  {
    double d = 0.0;
    for (int i = 0; i < dim; ++i) {
      d += grad[i] * p[i];
    }
    return d;
  }

  __device__ double
  generate_random_double(hiprandState* state, double lower, double upper)
  {
    return lower + (upper + (-lower)) * hiprand_uniform_double(state);
  }

  __global__ void
  setup_curand_states(hiprandState* states, uint64_t seed, int N)
  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
      return;
    hiprand_init(seed, idx, 0, &states[idx]);
  }

  __device__ double
  atomicMinDouble(double* addr, double val)
  {
    // reinterpret the address as 64‑bit unsigned
    unsigned long long* ptr = reinterpret_cast<unsigned long long*>(addr);
    unsigned long long old_bits = *ptr, assumed_bits;

    do {
      assumed_bits = old_bits;
      double old_val = __longlong_as_double(assumed_bits);
      // if the current value is already <= our candidate, nothing to do
      if (old_val <= val)
        break;
      // else try to swap in the new min value’s bit‐pattern
      unsigned long long new_bits = __double_as_longlong(val);
      old_bits = atomicCAS(ptr, assumed_bits, new_bits);
    } while (assumed_bits != old_bits);

    // return the previous minimum
    return __longlong_as_double(old_bits);
  }

} // end namespace util
