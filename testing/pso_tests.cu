#include "hip/hip_runtime.h"
// pso_tests.cu

#include <catch2/catch_all.hpp>
#include <catch2/matchers/catch_matchers_floating_point.hpp>
using Catch::Approx;

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <array>
#include <algorithm>

#include "utils.cuh"
#include "pso.cuh"
#include "fun.h"
#include "bfgs.cuh"

using namespace zeus;

// helper to copy device→host
template <typename T>
void
copyDevice(const T* dptr, T* hptr, size_t n)
{
  hipMemcpy(hptr, dptr, n * sizeof(T), hipMemcpyDeviceToHost);
}

TEST_CASE("pso::initKernel sets pBest & gBest for util::Rastrigin<2>",
          "[pso][init]")
{
  constexpr int N = 1, DIM = 2;
  const double lower = -5.0, upper = 5.0;
  const uint64_t seed = 42;

  // hiprand states
  float ms_rand;
  hiprandState* d_states = bfgs::initialize_states(N, int(seed), ms_rand);

  // allocate PSO buffers
  double *dX, *dV, *dPBestX, *dPBestVal, *dGBestX, *dGBestVal;
  hipMalloc(&dX, N * DIM * sizeof(double));
  hipMalloc(&dV, N * DIM * sizeof(double));
  hipMalloc(&dPBestX, N * DIM * sizeof(double));
  hipMalloc(&dPBestVal, N * sizeof(double));
  hipMalloc(&dGBestX, DIM * sizeof(double));
  hipMalloc(&dGBestVal, sizeof(double));
  {
    double inf = std::numeric_limits<double>::infinity();
    hipMemcpy(dGBestVal, &inf, sizeof(double), hipMemcpyHostToDevice);
  }

  // run initKernel<Function,DIM>
  pso::initKernel<util::Rastrigin<DIM>, DIM><<<1, N>>>(util::Rastrigin<DIM>(),
                                                       lower,
                                                       upper,
                                                       dX,
                                                       dV,
                                                       dPBestX,
                                                       dPBestVal,
                                                       dGBestX,
                                                       dGBestVal,
                                                       N,
                                                       seed,
                                                       d_states);
  hipDeviceSynchronize();

  // copy back
  double hPVal, hGVal;
  double hPX[DIM], hGX[DIM];
  copyDevice(dPBestVal, &hPVal, 1);
  copyDevice(dGBestVal, &hGVal, 1);
  copyDevice(dPBestX, hPX, DIM);
  copyDevice(dGBestX, hGX, DIM);

  // compute expected f(pBestX) on host
  std::array<double, DIM> arr;
  std::copy(hPX, hPX + DIM, arr.begin());
  double expected = util::Rastrigin<DIM>()(arr);

  REQUIRE(hPVal == Approx(expected).margin(1e-6));
  REQUIRE(hGVal == Approx(expected).margin(1e-6));
  for (int d = 0; d < DIM; ++d)
    REQUIRE(hGX[d] == Approx(hPX[d]).margin(1e-6));

  // cleanup
  hipFree(dX);
  hipFree(dV);
  hipFree(dPBestX);
  hipFree(dPBestVal);
  hipFree(dGBestX);
  hipFree(dGBestVal);
  hipFree(d_states);
}

TEST_CASE("pso::iterKernel inertia‐only updates X and V for 4 particles in 1D",
          "[pso][component][iter]")
{
  constexpr int N = 4, DIM = 1;
  const double lower = 0.0, upper = 1.0;
  const uint64_t seed = 0;

  // init hiprand states (not actually used when c1=c2=0)
  float ms_rand;
  hiprandState* d_states = bfgs::initialize_states(N, int(seed), ms_rand);

  // allocate everything
  double *dX, *dV, *dPBestX, *dPBestVal, *dGBestX, *dGBestVal;
  hipMalloc(&dX, N * DIM * sizeof(double));
  hipMalloc(&dV, N * DIM * sizeof(double));
  hipMalloc(&dPBestX, N * DIM * sizeof(double));
  hipMalloc(&dPBestVal, N * sizeof(double));
  hipMalloc(&dGBestX, DIM * sizeof(double));
  hipMalloc(&dGBestVal, sizeof(double));

  // host‐side initial values
  double hX[N] = {0.0, 1.0, 2.0, 3.0};
  double hV[N] = {10.0, 20.0, 30.0, 40.0};
  // personal & global bests start out large so they won't interfere
  double hPBX[N];
  std::copy(hX, hX + N, hPBX);
  double hPBV[N];
  std::fill(hPBV, hPBV + N, 1e6);
  double hGBX[DIM] = {0.0};
  double hGBV = 1e6;

  hipMemcpy(dX, hX, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dV, hV, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dPBestX, hPBX, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dPBestVal, hPBV, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dGBestX, hGBX, DIM * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dGBestVal, &hGBV, sizeof(double), hipMemcpyHostToDevice);

  // launch one iteration: w=0.5, c1=c2=0
  pso::iterKernel<util::Rastrigin<DIM>, DIM><<<1, N>>>(util::Rastrigin<DIM>(),
                                                       lower,
                                                       upper,
                                                       0.5, // inertia
                                                       0.0, // cognitive
                                                       0.0, // social
                                                       dX,
                                                       dV,
                                                       dPBestX,
                                                       dPBestVal,
                                                       dGBestX,
                                                       dGBestVal,
                                                       nullptr,
                                                       false,
                                                       N,
                                                       /*iter=*/0,
                                                       seed,
                                                       d_states);
  hipDeviceSynchronize();

  // 5) copy back and verify: v1 = 0.5*v0, x1 = x0 + v1
  double hX1[N], hV1[N];
  hipMemcpy(hX1, dX, N * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(hV1, dV, N * sizeof(double), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; ++i) {
    double expectedV = 0.5 * hV[i];
    double expectedX = hX[i] + expectedV;
    REQUIRE(hV1[i] == Approx(expectedV).margin(1e-12));
    REQUIRE(hX1[i] == Approx(expectedX).margin(1e-12));
  }

  // cleanup
  hipFree(dX);
  hipFree(dV);
  hipFree(dPBestX);
  hipFree(dPBestVal);
  hipFree(dGBestX);
  hipFree(dGBestVal);
  hipFree(d_states);
}

TEST_CASE("pso::iterKernel with zero w,c1,c2 leaves X unchanged and V zero",
          "[pso][iter]")
{
  constexpr int N = 1, DIM = 2;
  const double lower = -5.0, upper = 5.0;
  const uint64_t seed = 42;

  // hiprand states & buffers
  float ms_rand;
  hiprandState* d_states = bfgs::initialize_states(N, int(seed), ms_rand);

  double *dX, *dV, *dPBestX, *dPBestVal, *dGBestX, *dGBestVal;
  hipMalloc(&dX, N * DIM * sizeof(double));
  hipMalloc(&dV, N * DIM * sizeof(double));
  hipMalloc(&dPBestX, N * DIM * sizeof(double));
  hipMalloc(&dPBestVal, N * sizeof(double));
  hipMalloc(&dGBestX, DIM * sizeof(double));
  hipMalloc(&dGBestVal, sizeof(double));

  // init
  pso::initKernel<util::Rastrigin<DIM>, DIM><<<1, N>>>(util::Rastrigin<DIM>(),
                                                       lower,
                                                       upper,
                                                       dX,
                                                       dV,
                                                       dPBestX,
                                                       dPBestVal,
                                                       dGBestX,
                                                       dGBestVal,
                                                       N,
                                                       seed,
                                                       d_states);
  hipDeviceSynchronize();

  // 3) snapshot pre‐iter
  double hX0[DIM], hV0[DIM], hPB0[DIM], hGB0[DIM], hPV0, hGV0;
  copyDevice(dX, hX0, DIM);
  copyDevice(dV, hV0, DIM);
  copyDevice(dPBestX, hPB0, DIM);
  copyDevice(dGBestX, hGB0, DIM);
  copyDevice(dPBestVal, &hPV0, 1);
  copyDevice(dGBestVal, &hGV0, 1);

  // 4) one iteration with w=c1=c2=0
  pso::iterKernel<util::Rastrigin<DIM>, DIM><<<1, N>>>(util::Rastrigin<DIM>(),
                                                       lower,
                                                       upper,
                                                       0.0,
                                                       0.0,
                                                       0.0, // w, c1, c2
                                                       dX,
                                                       dV,
                                                       dPBestX,
                                                       dPBestVal,
                                                       dGBestX,
                                                       dGBestVal,
                                                       nullptr,
                                                       false,
                                                       N,
                                                       0,
                                                       seed,
                                                       d_states);
  hipDeviceSynchronize();

  // 5) snapshot post‐iter
  double hX1[DIM], hV1[DIM], hPB1[DIM], hGB1[DIM], hPV1, hGV1;
  copyDevice(dX, hX1, DIM);
  copyDevice(dV, hV1, DIM);
  copyDevice(dPBestX, hPB1, DIM);
  copyDevice(dGBestX, hGB1, DIM);
  copyDevice(dPBestVal, &hPV1, 1);
  copyDevice(dGBestVal, &hGV1, 1);

  // 6) assertions
  for (int d = 0; d < DIM; ++d) {
    REQUIRE(hX1[d] == Approx(hX0[d]).margin(1e-12));
    REQUIRE(hV1[d] == Approx(0.0).margin(1e-12));
    REQUIRE(hPB1[d] == Approx(hPB0[d]).margin(1e-12));
    REQUIRE(hGB1[d] == Approx(hGB0[d]).margin(1e-12));
  }
  REQUIRE(hPV1 == Approx(hPV0).margin(1e-12));
  REQUIRE(hGV1 == Approx(hGV0).margin(1e-12));

  // cleanup
  hipFree(dX);
  hipFree(dV);
  hipFree(dPBestX);
  hipFree(dPBestVal);
  hipFree(dGBestX);
  hipFree(dGBestVal);
  hipFree(d_states);
}

inline int pointerStatus(double* p)
{
    if (p == MALLOC_ERROR)  return 3;
    if (p == KERNEL_ERROR)  return 4;
    return 0;                               // success
}

TEST_CASE("pso::launch returns 4: KERNEL_ERROR when overflowing the memory")
{
    // exhaust almost all free memory on the device ──────────────
    std::vector<void*> scraps;
    size_t freeB  = 0, totalB = 0;
    hipMemGetInfo(&freeB, &totalB);

    // Keep at least 16 MiB so the runtime itself can breathe
    const size_t CHUNK = (freeB > 32ull << 20) ? (freeB - (16ull << 20)) : freeB / 2;

    while (true) {
        void* p = nullptr;
        if (hipMalloc(&p, CHUNK) != hipSuccess) break;
        scraps.push_back(p);
    }

    // now launch PSO with a 'normal' problem size
    using Fn  = util::Rosenbrock<2>;
    constexpr int DIM = 2;
    float ms0 = 0.0f, ms1 = 0.0f;
    hiprandState* states = nullptr;

    double* ptr = pso::launch<Fn, DIM>(/*PSO_ITER*/ 10,
                                       /*N*/         512,
                                       /*lower*/    -2.0,
                                       /*upper*/     2.0,
                                       ms0, ms1,
                                       42, states,
                                       Fn{});

    REQUIRE(ptr == KERNEL_ERROR);

    // clean up the scrap buffers so the rest of the suite runs ‐─
    for (void* p : scraps) hipFree(p);
}

TEST_CASE("pso::launch returns 3 (MALLOC_ERROR) when hipMalloc fails",
          "[pso][malloc-error]")
{
    using Fn  = util::Rosenbrock<2>;
    constexpr int DIM       = 2;
    constexpr int PSO_ITER  = 10;

    // pick an absurdly large N so that at least one hipMalloc fails
    const int N = std::numeric_limits<int>::max() / DIM;   // ≈1 G x DIM doubles

    float ms_init = 0.0f, ms_pso = 0.0f;
    hiprandState* states = nullptr;

    double* ptr = pso::launch<Fn, DIM>(PSO_ITER, N, -2.0, 2.0,
                                       ms_init, ms_pso, 42, states, Fn{});

    REQUIRE(pointerStatus(ptr) == 3);
}

TEST_CASE("pso::launch returns 4 (KERNEL_ERROR) when the kernel launch fails",
          "[pso][kernel-error]")
{
    using Fn  = util::Rosenbrock<2>;
    constexpr int DIM       = 2;
    constexpr int PSO_ITER  = 10;

    // This deliberately supplies no RNG state array.
    // the kernel dereferences it, so the first launch triggers an invalid-device-pointer
    // error which `pso::launch` must translate to KERNEL_ERROR.
    hiprandState* states = nullptr;

    float ms_init = 0.0f, ms_pso = 0.0f;
    const int N = 512;

    double* ptr = pso::launch<Fn, DIM>(PSO_ITER, N, -2.0, 2.0,
                                       ms_init, ms_pso, 99, states, Fn{});

    REQUIRE(pointerStatus(ptr) == 4);
}

