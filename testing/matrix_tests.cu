#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include <catch2/catch_all.hpp>
#include "matrix.cuh"
#include <hip/hip_runtime.h>
#include <vector>

// Square matrix host element access & dims tests
//   Create an NxN matrix, fill with values and verify rows()==cols()==N
//   and operator()(i,j) retrieving exactly what was stored
TEST_CASE("matrix:square host element access & dims", "[matrix][host][square]") {
  constexpr std::size_t N = 4;

  // squre NxN matrixi
  Matrix<double> m(N, N);

  // fill host buffer: m(i,j) = i*N + j
  // This yields the flattened sequence [0,1,2,3, 4,5,6,7, 8,9,10,11, 12,13,14,15]
  for (std::size_t i = 0; i < N; ++i) {
    for (std::size_t j = 0; j < N; ++j) {
      m(i,j) = static_cast<double>(i * N + j);
    }
  }

  // check that both dimensions are equal to N
  REQUIRE(m.rows() == N);
  REQUIRE(m.cols() == N);

  // verify each element comes back exactly as stored
  for (std::size_t i = 0; i < N; ++i) {
    for (std::size_t j = 0; j < N; ++j) {
      double expect = double(i * N + j);
      REQUIRE(m(i,j) == Catch::Approx(expect));
    }
  }
}

// host element access, rows(), cols()
TEST_CASE("matrix: not square host element access & dims", "[matrix][host]") {
  constexpr std::size_t R = 2, C = 3;
  Matrix<double> m(R,C);
  

  // fill on host  m(i,j) = 1 + i*C + j 
  // so that the flattened sequence is [1,2,3,4,5,6]
  for (std::size_t i = 0; i < R; ++i)
    for (std::size_t j = 0; j < C; ++j)
      m(i,j) = static_cast<double>(i * C + j + 1.0);

  // dim
  REQUIRE(m.rows() == R);
  REQUIRE(m.cols() == C);

  // host readback and check
  for (std::size_t i = 0; i < R; ++i) {
    for (std::size_t j = 0; j < C; ++j) {
      double expect = double(i * C + j + 1);
      REQUIRE(m(i,j) == Catch::Approx(expect));
    }
  }
}

// kernel to read back every entry with Matrix<double>::operator() from the device
template<int R,int C>
__global__ void matrix_device_access(Matrix<double> m, double* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < R*C) {
    int i = idx / C;
    int j = idx % C;
    out[idx] = m(i,j);
  }
}

TEST_CASE("matrix: syncHostToDevice and device data()", "[matrix][device]") {
  constexpr int R = 2, C = 3, N = R*C;
  Matrix<double> m(R,C);

  // fill host buffer
  for (int i = 0; i < R; ++i)
    for (int j = 0; j < C; ++j)
      m(i,j) = double(i * C + j + 10);

  // push to GPU
  m.syncHostToDevice();

  // allocate output
  double* d_out = nullptr;
  hipMalloc(&d_out, N * sizeof(double));
  // launch 1 block of N threads
  matrix_device_access<R,C><<<1, N>>>(m, d_out);
  REQUIRE(hipDeviceSynchronize() == hipSuccess);

  // copy back and check
  std::vector<double> host_out(N);
  hipMemcpy(host_out.data(), d_out, N*sizeof(double), hipMemcpyDeviceToHost);

  for (int idx = 0; idx < N; ++idx) {
    double expect = double(idx + 10);
    REQUIRE(host_out[idx] == Catch::Approx(expect));
  }
  hipFree(d_out);
}


